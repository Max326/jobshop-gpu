#include "hip/hip_runtime.h"
#include "customCMAES.hpp"
#include "JobShopGPUEvaluator.cuh"
#include <fstream> // Dodaj na górze pliku

/*
:0 will be used to mark OG Stanislaus's code regarding cmaes, 
so that i (or Copilot) dont fuck it up 
*/
using namespace libcmaes;

JobShopGPUEvaluator* g_gpu_train_evaluator = nullptr;
JobShopGPUEvaluator* g_gpu_validate_evaluator = nullptr;

float best_val_makespan = std::numeric_limits<float>::max();
Eigen::VectorXd best_weights; // Will hold the best weights found so far

int main(int argc, char *argv[])
{
    const std::vector<int> topology = {86, 32, 16, 1};
    const int batch_size = 50;
    const int train_problem_count = 130000; //130k
    const int validation_problem_count = 10000; // TODO change to 10k!!! when file is ready

    // --- Nowe pliki ---
    const std::string train_problem_file = "TRAIN/rnd_JT(5)_J(15)_M(5)_JO(5-10)_O(20)_OM(1-3)_total.json"; // used to be test_problem_file
    const std::string validate_problem_file = "VALID/rnd_JT(5)_J(15)_M(5)_JO(5-10)_O(20)_OM(1-3)_validation.json";
    const std::string test_problem_file = "TEST/rnd_JT(5)_J(15)_M(5)_JO(5-10)_O(20)_OM(1-3)_test.json"; 

    int population_size = 192;//:0

    int nn_weights_count = NeuralNetwork::CalculateTotalParameters(topology);//:0
    
    std::cout<< nn_weights_count <<std::endl;
    
    JobShopGPUEvaluator gpu_evaluator(train_problem_file, topology, population_size, train_problem_count);
    g_gpu_train_evaluator = &gpu_evaluator;

    g_gpu_validate_evaluator = new JobShopGPUEvaluator(validate_problem_file, topology, population_size, validation_problem_count);

    // int total_problems = gpu_evaluator.GetTotalProblems();
    std::vector<double> x0(nn_weights_count, 0.0);
    /*     for(int i = 0; i < nn_weights_count; i++) {
        x0[i] = (double)rand() / RAND_MAX * 0.01 - 0.005;
        
    } */
    
    double sigma = 0.1;//:0
    CMAParameters<> cmaparams(x0, sigma, population_size);//:0  
    cmaparams.set_sep();
    cmaparams.set_algo(sepaCMAES);
    
    FitFunc eval = [](const double *x, const int N) -> double { return 0.0; }; //:0
    ESOptimizer<customCMAStrategy,CMAParameters<>> optim(eval, cmaparams);//:0

    // --- Dodaj pliki do zapisu najlepszych makespanów ---
    std::ofstream train_makespan_file("best_train_makespans.csv");
    std::ofstream val_makespan_file("best_val_makespans.csv");
    train_makespan_file << "iteration,best_train_makespan\n";
    val_makespan_file << "iteration,best_val_makespan\n";
    // ----------------------------------------------------

    int batch_start = 0;
    int global_iter=0;

    while(!optim.stop() && gpu_evaluator.SetCurrentBatch(batch_start, batch_size)) {
        dMat candidates = optim.ask();//:0
        optim.eval(candidates);//:0
        optim.tell();//:0
        optim.inc_iter();//:0

        // best_val_makespan = min(best_val_makespan, optim.get_best_fvalue()); //?
        std::cout << "Best makespan: " << optim.get_best_fvalue() << std::endl;

        // --- Zapisuj najlepszy makespan z treningu po każdej iteracji ---
        train_makespan_file << global_iter << "," << optim.get_best_fvalue() << "\n";
        train_makespan_file.flush();
        // ---------------------------------------------------------------

        batch_start += batch_size;
        global_iter++;
        
        std::cout << "Global iterations: " << global_iter << std::endl;

        if (global_iter % 10 == 0) {
            // 1. Get the best weights from the current training population
            const auto& cma_weights = optim.get_solutions().best_candidate().get_x();
            Eigen::VectorXd current_best_weights = Eigen::Map<const Eigen::VectorXd>(cma_weights.data(), cma_weights.size());

            // 2. Evaluate this candidate on 10,000 problems to get the lowest makespan
            std::cout << "[VALIDATION] Iter " << global_iter 
                    << ": Running validation on " << validation_problem_count << " problems..." << std::endl;

            // const int validation_batch_size = validation_problem_count / population_size;
                    
            float lowest_makespan = g_gpu_validate_evaluator->EvaluateForMinMakespan(current_best_weights, validation_problem_count);

            std::cout << "[VALIDATION] Lowest makespan found = " << lowest_makespan
                    << " (best so far: " << best_val_makespan << ")" << std::endl;

            // --- Zapisuj najlepszy makespan z walidacji po każdej iteracji walidującej ---
            val_makespan_file << global_iter << "," << lowest_makespan << "\n";
            val_makespan_file.flush();
            // ------------------------------------------------------------------------------

            // 3. If it's a new global best, save the weights
            if (lowest_makespan < best_val_makespan) {
                std::cout << "[VALIDATION] New best network found!" << std::endl;
                best_val_makespan = lowest_makespan;
                best_weights = current_best_weights;

                // --- Your existing file-saving code ---
                try {
                    const std::filesystem::path weights_path("best_weights.csv");
                    std::filesystem::path dir_path = weights_path.parent_path();
                    if (!dir_path.empty()) {
                        std::filesystem::create_directories(dir_path);
                    }
                    std::cout << "[IO] Saving new best weights to: " << weights_path << std::endl;
                    std::ofstream file(weights_path);
                    if (file.is_open()) {
                        const static Eigen::IOFormat CSVFormat(Eigen::StreamPrecision, Eigen::DontAlignCols, ", ", "\n");
                        file << best_weights.transpose().format(CSVFormat);
                        file.close();
                    } else {
                        std::cerr << "[ERROR] Unable to open file for writing: " << weights_path << std::endl;
                    }
                } catch (const std::filesystem::filesystem_error& e) {
                    std::cerr << "[ERROR] Filesystem error: " << e.what() << std::endl;
                }
                // --- End of file-saving code ---
            }
        }
        
    }

    // --- Zamknij pliki na końcu programu ---
    train_makespan_file.close();
    val_makespan_file.close();
    // ----------------------------------------

    // --- TEST NA ZBIORZE TESTOWYM ---
    try {
        const int test_problem_count = 100; // lub inna liczba, jeśli masz inny rozmiar testu

        JobShopGPUEvaluator test_evaluator(test_problem_file, topology, population_size, test_problem_count);

        std::cout << "[TEST] Evaluating best weights on test set (" << test_problem_count << " problems)..." << std::endl;
        float test_makespan = test_evaluator.EvaluateForMinMakespan(best_weights, test_problem_count);

        // Zapisz wynik do pliku
        std::ofstream test_result_file("best_test_result.csv");
        if (test_result_file.is_open()) {
            test_result_file << "makespan," << test_makespan << "\n";
            test_result_file << "weights,";
            for (int i = 0; i < best_weights.size(); ++i) {
                test_result_file << best_weights[i];
                if (i < best_weights.size() - 1) test_result_file << ",";
            }
            test_result_file << "\n";
            test_result_file.close();
            std::cout << "[TEST] Test result saved to best_test_result.csv" << std::endl;
        } else {
            std::cerr << "[ERROR] Unable to open best_test_result.csv for writing!" << std::endl;
        }
    } catch (const std::exception& e) {
        std::cerr << "[ERROR][TEST] Exception during test evaluation: " << e.what() << std::endl;
    }
    // --- KONIEC TESTU ---

    return 0;
}