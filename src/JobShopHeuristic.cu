#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <nlohmann/json.hpp>

#include "JobShopHeuristic.cuh"

using json = nlohmann::json;

// Constructors
JobShopHeuristic::JobShopHeuristic(const std::vector<int>& topology)
	: neuralNetwork(topology) {}

JobShopHeuristic::JobShopHeuristic(NeuralNetwork&& net)
	: neuralNetwork(std::move(net)) {}

// Copy solution from GPU to CPU
void JobShopHeuristic::CPUSolution::FromGPU(const SolutionManager::GPUSolutions& gpuSols, int problemId) {
	int counts_offset = problemId * gpuSols.numMachines;
	int schedule_offset = problemId * gpuSols.numMachines * gpuSols.maxOps;

	std::vector<int> counts(gpuSols.numMachines);
	hipMemcpy(counts.data(), gpuSols.allScheduleCounts + counts_offset,
			   sizeof(int) * gpuSols.numMachines, hipMemcpyDeviceToHost);

	hipMemcpy(&makespan, gpuSols.allMakespans + problemId,
			   sizeof(int), hipMemcpyDeviceToHost);

	std::vector<OperationSchedule> allOps(gpuSols.numMachines * gpuSols.maxOps);
	hipMemcpy(allOps.data(), gpuSols.allSchedules + schedule_offset,
			   sizeof(OperationSchedule) * allOps.size(), hipMemcpyDeviceToHost);

	schedule.resize(gpuSols.numMachines);
	for(int m = 0; m < gpuSols.numMachines; ++m) {
		schedule[m].clear();
		for(int i = 0; i < counts[m] && i < gpuSols.maxOps; ++i) {
			int idx = m * gpuSols.maxOps + i;
			schedule[m].push_back(allOps[idx]);
		}
	}
}

// Copy solution from CPU to GPU
SolutionManager::GPUSolutions JobShopHeuristic::CPUSolution::ToGPU() const {
	SolutionManager::GPUSolutions gpuSol;
	gpuSol.numMachines = schedule.size();

	hipMalloc(&gpuSol.allSchedules, sizeof(OperationSchedule) * schedule.size() * MAX_OPS);
	hipMalloc(&gpuSol.allScheduleCounts, sizeof(int) * schedule.size());
	hipMalloc(&gpuSol.allMakespans, sizeof(int));

	hipMemcpy(gpuSol.allMakespans, &makespan, sizeof(int), hipMemcpyHostToDevice);

	std::vector<OperationSchedule> flat_schedule;
	std::vector<int> counts;
	for(const auto& machine: schedule) {
		flat_schedule.insert(flat_schedule.end(), machine.begin(), machine.end());
		counts.push_back(machine.size());
	}

	hipMemcpy(gpuSol.allSchedules, flat_schedule.data(),
			   sizeof(OperationSchedule) * flat_schedule.size(),
			   hipMemcpyHostToDevice);
	hipMemcpy(gpuSol.allScheduleCounts, counts.data(),
			   sizeof(int) * counts.size(),
			   hipMemcpyHostToDevice);

	return gpuSol;
}

// TODO one stream, not new ones

void JobShopHeuristic::SolveBatchNew(
	const GPUProblem* problems,
	const NeuralNetwork::DeviceEvaluator* evaluators,
	GPUOperation* ops_working,
	float* results,
	int numProblems_per_block,		// num FJSS problems this block will handle
	int numWeights_total_blocks, 	// total NNs, so total blocks
	int numWeights_per_block,  		// how many weights per block
	int numBiases_per_block, 		// how many biases per block
	int maxOpsPerProblem,
	hipStream_t stream,			// Removed default stream = 0 as it's passed from evaluator
	int nn_total_params_for_one_network,
	bool validation_mode
) {
	int threads_per_block = 192;	 // This is your blockDim.x
	int total_cuda_blocks = numWeights_total_blocks;

	// Calculate dynamic shared memory size:
	// (threads_per_block * sizeof(float) for shared_makespans)
	// + (nn_total_params_for_one_network * sizeof(float) for combined weights & biases of ONE network)
	size_t dynamic_shared_mem_size = (threads_per_block * sizeof(float)) + (nn_total_params_for_one_network * sizeof(float));

	// hipDeviceProp_t deviceProp;
	// hipGetDeviceProperties(&deviceProp, 0);
	// size_t maxSharedMemoryPerBlock = deviceProp.sharedMemPerBlock;

	// if (dynamic_shared_mem_size > maxSharedMemoryPerBlock) {
	// 	printf("ERROR: Requested shared memory (%zu) exceeds maximum (%zu)\n", dynamic_shared_mem_size, maxSharedMemoryPerBlock);
	// 	return;
	// }

	hipDeviceSetLimit(hipLimitStackSize, 4096);
	// int reset_value = 0; // If gpu_error_flag is used
	// hipMemcpyToSymbol(HIP_SYMBOL(gpu_error_flag), &reset_value, sizeof(int), 0, hipMemcpyHostToDevice);

	__launch_bounds__(192, 4) SolveManyWeightsKernel<<<total_cuda_blocks, threads_per_block, dynamic_shared_mem_size, stream>>>(
		problems,
		evaluators,
		ops_working,
		results,
		numProblems_per_block,	// This is how many problems each block should iterate up to.
		numWeights_per_block,	
		numBiases_per_block,
		maxOpsPerProblem,
		validation_mode);

	hipDeviceSynchronize();
}

// Allocate GPU memory for solutions
SolutionManager::GPUSolutions SolutionManager::CreateGPUSolutions(int numProblems, int numMachines, int maxOps) {
	GPUSolutions solutions;
	solutions.numProblems = numProblems;
	solutions.numMachines = numMachines;
	solutions.maxOps = maxOps;

	size_t schedule_size = sizeof(OperationSchedule) * numMachines * maxOps * numProblems;
	hipMalloc(&solutions.allSchedules, schedule_size);
	hipMemset(solutions.allSchedules, 0, schedule_size);

	size_t counts_size = numProblems * numMachines * sizeof(int);
	hipMalloc(&solutions.allScheduleCounts, counts_size);
	hipMemset(solutions.allScheduleCounts, 0, counts_size);

	hipMalloc(&solutions.allMakespans, sizeof(int) * numProblems);
	hipMemset(solutions.allMakespans, 0, numProblems * sizeof(int));

	return solutions;
}

// Free GPU memory for solutions
void SolutionManager::FreeGPUSolutions(SolutionManager::GPUSolutions& sols) {
	hipFree(sols.allSchedules);
	hipFree(sols.allScheduleCounts);
	hipFree(sols.allMakespans);
	sols = GPUSolutions {};
}

// Print schedule for a solution
void JobShopHeuristic::PrintSchedule(const CPUSolution& solution, JobShopData data) {
	// Build machine->operation types map if not already available
	if(data.machineEligibleOperations.empty()) {
		data.BuildMachineEligibleOperations();
	}

	std::cout << "\n=== FINAL SCHEDULE ===" << std::endl;

	for(int machineId = 0; machineId < solution.schedule.size(); ++machineId) {
		std::cout << "M" << machineId << " (";
		bool firstOp = true;
		for(int opType: data.machineEligibleOperations[machineId]) {
			if(!firstOp) std::cout << ", ";
			std::cout << opType;
			firstOp = false;
		}
		std::cout << "): [";

		int currentTime = 0;
		bool firstElement = true;
		const auto& machineSchedule = solution.schedule[machineId];

		if(machineSchedule.empty()) {
			std::cout << "idle";
		} else {
			for(const auto& scheduledOp: machineSchedule) {
				if(scheduledOp.startTime > currentTime) {
					if(!firstElement) std::cout << "][";
					std::cout << "w-" << (scheduledOp.startTime - currentTime);
					firstElement = false;
					currentTime = scheduledOp.startTime;
				}
				if(!firstElement) std::cout << "][";
				std::cout << "t=" << scheduledOp.startTime << ",j" << scheduledOp.jobId
						  << "-o" << scheduledOp.opType
						  << "-" << (scheduledOp.endTime - scheduledOp.startTime);
				currentTime = scheduledOp.endTime;
				firstElement = false;
			}
		}
		std::cout << "]" << std::endl;
	}
	std::cout << "Makespan: " << solution.makespan << std::endl;
}

// Update schedule after scheduling an operation (obsolete)
void JobShopHeuristic::UpdateSchedule(JobShopData& data, int jobId, int operationIdx,
									  int machineId, CPUSolution& solution) {
	auto& job = data.jobs[jobId];
	const auto& operation = job.operations[operationIdx];

	int processingTime = data.processingTimes[operation.type][machineId];
	if(processingTime <= 0) {
		std::cerr << "Warning: Attempted to schedule zero-duration operation\n";
		return;
	}

	int machineAvailableTime = solution.schedule[machineId].empty()
								   ? 0
								   : solution.schedule[machineId].back().endTime;

	int startTime = std::max(machineAvailableTime, job.lastOpEndTime);
	int endTime = startTime + processingTime;

	solution.schedule[machineId].push_back({jobId, operation.type, startTime, endTime});
	job.lastOpEndTime = endTime;
	job.nextOpIndex++;
	solution.makespan = std::max(solution.makespan, endTime);
}

__global__ __launch_bounds__(192, 4) void SolveManyWeightsKernel(
	const GPUProblem* problems,
	const NeuralNetwork::DeviceEvaluator* evaluators,  // This points to DeviceEvaluators in global memory
	GPUOperation* ops_working,
	float* results,
	int total_problems_in_batch,	 // Renamed for clarity (was numProblems)
	int numWeights_per_block,	
	int	numBiases_per_block,
	int maxOpsPerProblem,
	bool validation_mode) {

	// Combined dynamic shared memory
	extern __shared__ float shared_block_data[];

	// Partition 1: Makespans for each problem solved by threads in this block
	// blockDim.x is the number of threads in this block (e.g., 64)
	float* shared_makespans = shared_block_data;

	// --- Identify current weight set and problem for this thread ---
	int weightSet = blockIdx.x;			  // Each block handles one weightSet
	int problemIdxInBlock = threadIdx.x;  // Each thread in block handles one FJSS problem for this weightSet

	// --- Load NN Parameters into Shared Memory ---
	const NeuralNetwork::DeviceEvaluator& nn_eval_global_ptr = evaluators[weightSet];  // Get the evaluator for this block

	// Partition 2: Storage for NN weights for this block (starts after shared_makespans)
	float* sm_weights = shared_block_data + blockDim.x;

	// Partition 3: Storage for NN biases for this block (starts after sm_weights)
	float* sm_biases = shared_block_data + blockDim.x + numWeights_per_block;
	
	// Load weights cooperatively and coalesced
	// Calculate how many elements each thread might load in total passes
	int num_passes_weights = (numWeights_per_block + blockDim.x - 1) / blockDim.x;
	for (int pass = 0; pass < num_passes_weights; ++pass) {
		int current_element_idx = pass * blockDim.x + threadIdx.x;
		if (current_element_idx < numWeights_per_block) {
			sm_weights[current_element_idx] = nn_eval_global_ptr.weights[current_element_idx];
		}
	}

	// Load biases cooperatively and coalesced
	int num_passes_biases = (numBiases_per_block + blockDim.x - 1) / blockDim.x;
	for (int pass = 0; pass < num_passes_biases; ++pass) {
		int current_element_idx = pass * blockDim.x + threadIdx.x;
		if (current_element_idx < numBiases_per_block) {
			sm_biases[current_element_idx] = nn_eval_global_ptr.biases[current_element_idx];
		}
	}

	__syncthreads();  // IMPORTANT: Ensure all threads finish loading before any thread proceeds

	int problem_idx_to_solve = -1;

	if (validation_mode) {
        // --- VALIDATION PATH: Grid-wide indexing ---
        // Each thread across the entire grid gets a unique problem from the large 10k batch.
        int global_thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_thread_idx < total_problems_in_batch) {
            problem_idx_to_solve = global_thread_idx;
        }
    } else {
        // --- TRAINING PATH: Original block-local indexing ---
        // Each thread solves a problem from the small batch of 50.
        // All blocks solve the SAME set of 50 problems.
        int problemIdxInBlock = threadIdx.x;
        if (problemIdxInBlock < total_problems_in_batch) {
            problem_idx_to_solve = problemIdxInBlock;
        }
    }

	// --- Main problem-solving logic ---
	float makespan_val = 0.0f;	// Changed variable name to avoid conflict
	if(problem_idx_to_solve != -1) {
		const GPUProblem problem = problems[problem_idx_to_solve];	 // Assuming 'problems' array is correctly indexed for the batch

		// local_ops indexing seems correct from your previous structure
		// const int base_op_idx = (weightSet * total_problems_in_batch + problem_idx_to_solve) * maxOpsPerProblem;

		int base_op_idx;
		if (validation_mode) {
			// VALIDATION: The 'ops_working' buffer contains 192 copies of the problem set.
			// We will only use the FIRST copy (the one for candidate 0).
			// The index is based ONLY on the globally unique problem ID this thread is solving.
			base_op_idx = problem_idx_to_solve * maxOpsPerProblem; // TODO check if this is correct
		} else {
			// TRAINING: The original logic is correct here. Each block (weightSet) uses its
			// own distinct segment of the buffer for the 50-problem training batch.
			base_op_idx = (weightSet * total_problems_in_batch + problem_idx_to_solve) * maxOpsPerProblem;
		}

		GPUOperation* local_ops = &ops_working[base_op_idx];

		unsigned short int unscheduledOps = 0; // validation

		unsigned short int jobScheduledOps[MAX_JOBS] = {0};
		unsigned short int machine_times[MAX_MACHINES] = {0};

		unsigned short int jobTypeCount[MAX_JOB_TYPES] = {0};
		unsigned short int opTypeCount[MAX_OP_TYPES] = {0};
		unsigned short int opTypePerJobCount[MAX_JOBS][MAX_OP_TYPES] = {0};

		const int numJobs = problem.numJobs;
		const int numMachines = problem.numMachines;

		for(int jobID = 0; jobID < numJobs; ++jobID) {
			const GPUJob& job = problem.jobs[jobID];
			jobTypeCount[job.type]++;
			for(int opID = 0; opID < job.operationCount; ++opID) {
				GPUOperation& op = local_ops[job.operationsOffset + opID];
				opTypePerJobCount[jobID][op.type]++;
				opTypeCount[op.type]++;
				unscheduledOps++;
			}
		}

		
		int current_local_makespan = 0;
		bool scheduled_any;
		do {
			scheduled_any = false;
			float bestScoreValue = -FLT_MAX;
			int bestJobID = -1, bestOpID = -1, bestMachineID = -1;
			int bestStartTime = 0;

			for(int jobID = 0; jobID < numJobs; ++jobID) {
				if(jobScheduledOps[jobID] == problem.jobs[jobID].operationCount)
					continue;

				GPUJob& job = problem.jobs[jobID];

				for(int operationID = 0; operationID < job.operationCount; ++operationID) {
					GPUOperation& operation = local_ops[job.operationsOffset + operationID];
					if(operation.predecessorCount != 0) continue;

					for(int m = 0; m < operation.eligibleCount; m++) {
						int machineID = problem.eligibleMachines[operation.eligibleMachinesOffset + m];
						int start_time = max(machine_times[machineID], operation.lastPredecessorEndTime);
						int opMach_idx = operation.type * numMachines + machineID;
						int pTime = problem.processingTimes[opMach_idx];

						// Debug: op details
						/*                         if (weightSet == 0 && problemIdx == 0 && jobID == 0 && operationID == 0) {
													printf("[KERNEL] Operation details: jobID=%d, opID=%d, machineID=%d, start_time=%d, pTime=%d\n",
														  jobID, operationID, machineID, start_time, pTime);
												}
						 */
						float features[1 + 2 * MAX_MACHINES + 3 * MAX_OP_TYPES + MAX_JOB_TYPES] = {0.0f}; // TODO feature number

						int startIndex = 0;

						features[startIndex++] = ScaleTanh2(static_cast<float>(start_time) - machine_times[machineID]);  // wasted time

						for(int i = 0; i < MAX_MACHINES; ++i) {
							features[startIndex + i] = ScaleTanh2(static_cast<float>(current_local_makespan - machine_times[i]));  // envelope
						}
						features[startIndex + machineID] = ScaleTanh2(static_cast<float>(current_local_makespan - (start_time + pTime)));  // envelope for current machine
						
						startIndex += MAX_MACHINES;

						features[startIndex + machineID] = 1.0f;			 // one hot machine encoding
						startIndex += MAX_MACHINES;

						features[startIndex + operation.type] = 1.0f;	 // one hot operation type encoding
						startIndex += MAX_OP_TYPES;

						//* total number of operations left (of each type) - start
                        for (int i = 0; i < MAX_OP_TYPES; i++){
                            features[startIndex + i] = ScaleTanh2(static_cast<float>(opTypeCount[i]));
                        }
						features[startIndex + operation.type] = ScaleTanh2(static_cast<float>(opTypeCount[operation.type] - 1));


						// --features[totOpLeftStart + operation.type]; // because we score for the operation as if it was processed 
                        //* total number of operations left (of each type) - end

                        //* job's operations left (of each type) - start
                        startIndex += MAX_OP_TYPES;
                        for (int i = 0; i < MAX_OP_TYPES; i++){
                            features[startIndex + i] = ScaleTanh2(static_cast<float>(opTypePerJobCount[jobID][i]));
                        }
						features[startIndex + operation.type] = ScaleTanh2(static_cast<float>(opTypePerJobCount[jobID][operation.type] - 1));
                        // --features[jobOpLeftStart + operation.type]; // because we score for the operation as if it was processed
                        //* job's operations left (of each type) - end

                        //* one hot job type encoding - start
                        startIndex += MAX_OP_TYPES;
                        features[startIndex + job.type] = 1.0f; // one hot job type encoding
                        //* one hot job type encoding - end

						// TODO? operations left for job types (of each type)

                        //* total number of jobs left (of each type) - start
                        // int jobTypeCountStart = 1 + 2* MAX_MACHINES + 3 * MAX_OP_TYPES + MAX_JOB_TYPES;
                        // for (int i = jobTypeCountStart; i < jobTypeCountStart + MAX_JOB_TYPES; i++){
                        //     features[i] = static_cast<float>(jobTypeCount[i-jobTypeCountStart]);
                        // }
                        // --features[jobTypeCountStart + job.type]; // because we score for the operation as if it was processed
                        //* total number of jobs left (of each type) - end


						float score = nn_eval_global_ptr.Evaluate(features, sm_weights, sm_biases);

						if(score > bestScoreValue) {
							bestScoreValue = score;
							bestJobID = jobID;
							bestOpID = operationID;
							bestMachineID = machineID;
							bestStartTime = start_time;
						}
					}
				}
			}

			if(bestJobID == -1) break;

			// Debug: Score print
			// if(weightSet == 0 && problem_idx_to_solve == 0 && threadIdx.x == 0 && bestJobID == 0 && bestOpID == 0) {
			// 	printf("[DEBUG] Initial Score=%.2f\n", bestScoreValue);
			// }

			GPUJob& bestJob = problem.jobs[bestJobID];	// problem is const, so bestJob needs to be const if problem.jobs not modifiable
			GPUOperation& bestOperation = local_ops[bestJob.operationsOffset + bestOpID];
			int opMach_idx = bestOperation.type * problem.numMachines + bestMachineID;	// Use problem.numMachines
			int pTime = problem.processingTimes[opMach_idx];
			int endTime = bestStartTime + pTime;

			jobScheduledOps[bestJobID]++;
			opTypePerJobCount[bestJobID][bestOperation.type]--;
			opTypeCount[bestOperation.type]--;

			if(jobScheduledOps[bestJobID] == bestJob.operationCount) {
				jobTypeCount[bestJob.type]--;
			}

			unscheduledOps--;
			
			bestOperation.predecessorCount = -1;
			for(int s = 0; s < bestOperation.successorCount; ++s) {
				int successor_op_array_idx = problem.successorsIDs[bestOperation.successorsOffset + s];
				// The successorID is an index relative to the start of the current job's operations.
				GPUOperation& successorOperation = local_ops[bestJob.operationsOffset + successor_op_array_idx];
				successorOperation.predecessorCount -= 1;
				successorOperation.lastPredecessorEndTime = max(successorOperation.lastPredecessorEndTime, endTime);
			}
			machine_times[bestMachineID] = endTime;
			if(endTime > current_local_makespan) current_local_makespan = endTime;
			scheduled_any = true;
		} while(scheduled_any);

		makespan_val = static_cast<float>(current_local_makespan);
		
		if (unscheduledOps != 0) {
			// Debug: Print problem details if there are unscheduled operations
			printf("[KERNEL] Unscheduled operations remaining: %d\n", unscheduledOps);
		}
	} 
	
	shared_makespans[threadIdx.x] = makespan_val;

	// else {
	// 	// Threads outside the numProblemsToSolvePerBlock range (e.g. if blockDim.x > numProblemsToSolvePerBlock)
	// 	shared_makespans[threadIdx.x] = 0.0f;
	// }

	__syncthreads();

	// Reduction to calculate average makespan for this weightSet (block)
	if (threadIdx.x == 0) {
        if (validation_mode) {
            // VALIDATION: Find the minimum makespan among problems solved by this block's threads
            float min_makespan = FLT_MAX;
            int problems_in_this_block = 0;
            if (blockIdx.x < gridDim.x - 1) {
                problems_in_this_block = blockDim.x;
            } else {
                problems_in_this_block = total_problems_in_batch - (blockIdx.x * blockDim.x);
            }

            for (int i = 0; i < problems_in_this_block; ++i) {
                if (shared_makespans[i] > 0.0f) { // Only consider valid makespans
                    min_makespan = fminf(min_makespan, shared_makespans[i]);
                }
            }
            // results[weightSet] = (min_makespan == FLT_MAX) ? 0.0f : min_makespan;
			results[weightSet] = min_makespan;

        } else {
            // TRAINING: Calculate the average makespan over the 50 problems
            float sum = 0.0f;
            // total_problems_in_batch will be 50 here.
            for (int i = 0; i < total_problems_in_batch; ++i) {
                sum += shared_makespans[i];
            }
            results[weightSet] = (total_problems_in_batch > 0) ? (sum / total_problems_in_batch) : 0.0f;
        }
    }
}

// Print problem details from device (for debugging)
__device__ void PrintProblemDetails(const GPUProblem& problem) {
	printf("\n=== Problem %d Details ===\n", blockIdx.x * blockDim.x + threadIdx.x);
	printf("Machines: %d, Jobs: %d, Operation Types: %d\n",
		   problem.numMachines, problem.numJobs, problem.numOpTypes);

	printf("\nJobs:\n");
	for(int j = 0; j < problem.numJobs; j++) {
		GPUJob job = problem.jobs[j];
		printf("Job %d, of type %d (%d ops):\n", job.id, job.type, job.operationCount);

		for(int o = 0; o < job.operationCount; o++) {
			GPUOperation op = problem.operations[job.operationsOffset + o];
			printf("  Op type %d on machines: ", op.type);

			for(int m = 0; m < op.eligibleCount; m++) {
				printf("%d ", problem.eligibleMachines[op.eligibleMachinesOffset + m]);
			}
			printf("\n");
		}
	}
	printf("========================\n\n");
}