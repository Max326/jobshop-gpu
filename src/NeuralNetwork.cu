#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cmath>
#include <cstdlib>
#include <ctime>
#include <fstream>

#include "NeuralNetwork.cuh"

__device__ __managed__ int gpu_error_flag = 0;

void NeuralNetwork::InitializeCudaData() {
	// 1. Calculate offsets for each layer's weights and biases
	FlattenParams();

	/* std::cout << "\nFlattened weights size: " << flattenedWeights.size() << "\n";
	std::cout << "Flattened biases size: " << flattenedBiases.size() << "\n"; */

	/* std::cout << "First few weights: ";
	for(int i = 0; i < std::min(5, (int)flattenedWeights.size()); i++)
		std::cout << flattenedWeights[i] << " ";
		std::cout << "\n"; */

	layerOffsets.resize(weights.size());
	biasOffsets.resize(biases.size());
	size_t total_weights = 0;
	size_t total_biases = 0;

	for(size_t i = 0; i < weights.size(); ++i) {
		layerOffsets[i] = total_weights;
		total_weights += weights[i].size();

		biasOffsets[i] = total_biases;
		total_biases += biases[i].size();
	}

	// 2. Allocate GPU memory for weights and biases
	CUDA_CHECK(hipMalloc(&cudaData->d_weights, total_weights * sizeof(float)));
	CUDA_CHECK(hipMalloc(&cudaData->d_biases, total_biases * sizeof(float)));

	// 3. Find the maximum layer size for input/output buffers
	int max_layer_size = 0;
	for(int size: topology) {
		if(size > max_layer_size) {
			max_layer_size = size;
		}
	}

	// Allocate input and output buffers to the maximum layer size
	CUDA_CHECK(hipMalloc(&cudaData->d_input, max_layer_size * sizeof(float)));
	CUDA_CHECK(hipMalloc(&cudaData->d_output, max_layer_size * sizeof(float)));

	// 4. Initialize weight buffers to zero
	CUDA_CHECK(hipMemset(cudaData->d_weights, 0.0f, total_weights * sizeof(float)));
	CUDA_CHECK(hipMemset(cudaData->d_biases, 0.0f, total_biases * sizeof(float)));

	// 5. Copy weights and biases to GPU
	size_t weight_offset = 0;
	size_t bias_offset = 0;

	for(size_t i = 0; i < weights.size(); ++i) {
		CUDA_CHECK(hipMemcpy(cudaData->d_weights + weight_offset,
							  weights[i].data(),
							  weights[i].size() * sizeof(float),
							  hipMemcpyHostToDevice));

		CUDA_CHECK(hipMemcpy(cudaData->d_biases + bias_offset,
							  biases[i].data(),
							  biases[i].size() * sizeof(float),
							  hipMemcpyHostToDevice));

		weight_offset += weights[i].size();
		bias_offset += biases[i].size();
	}
}

NeuralNetwork::NeuralNetwork() : cudaData(std::make_unique<CudaData>()) {}

NeuralNetwork::NeuralNetwork(const std::vector<int> &topology,
							 const std::vector<std::vector<float>> *weights_ptr,
							 const std::vector<std::vector<float>> *biases_ptr)
	: topology(topology),
	  weights(weights_ptr ? *weights_ptr : std::vector<std::vector<float>>()),
	  biases(biases_ptr ? *biases_ptr : std::vector<std::vector<float>>()),
	  cudaData(std::make_unique<CudaData>()) {
	// ======================
	//  Validation checks
	// ======================

	if(weights_ptr == nullptr) {
		GenerateWeights();
	}
	if(biases_ptr == nullptr) {
		GenerateBiases();
	}

	// 1. Validate topology
	if(topology.empty()) {
		throw std::invalid_argument("NeuralNetwork: Topology cannot be empty");
	}

	if(topology.size() < 2) {
		throw std::invalid_argument("NeuralNetwork: Topology must have at least 2 layers (input/output)");
	}

	Validate();
	InitializeCudaData();
}

NeuralNetwork::~NeuralNetwork() {
	if(cudaData) {
		hipFree(cudaData->d_weights);
		hipFree(cudaData->d_biases);
		hipFree(cudaData->d_input);
		hipFree(cudaData->d_output);
	}
}

// Konstruktor przenoszący
NeuralNetwork::NeuralNetwork(NeuralNetwork &&other) noexcept
	: topology(std::move(other.topology)),
	  weights(std::move(other.weights)),
	  biases(std::move(other.biases)),
	  layerOffsets(std::move(other.layerOffsets)),
	  biasOffsets(std::move(other.biasOffsets)),
	  cudaData(std::move(other.cudaData)) {
	// Zabezpieczenie przed podwójnym zwolnieniem pamięci
	other.cudaData.reset(nullptr);
}

// Operator przypisania przenoszącego
NeuralNetwork &NeuralNetwork::operator=(NeuralNetwork &&other) noexcept {
	if(this != &other) {
		topology = std::move(other.topology);
		weights = std::move(other.weights);
		biases = std::move(other.biases);
		layerOffsets = std::move(other.layerOffsets);
		biasOffsets = std::move(other.biasOffsets);
		cudaData = std::move(other.cudaData);
	}
	return *this;
}

// Funkcja aktywacji scaleTanh2
__device__ float ScaleTanh2(float x) {
	// Sprawdź, czy wejście jest NaN lub Inf
	if(isnan(x) || isinf(x)) {
		printf("[ERROR] ScaleTanh2 received invalid input: %f\n", x);
		return 0.0f;
	}

	constexpr float shift = 3.5f;
	constexpr float rshift = 1.0f / shift;
	if(x >= 0.f) {
		if(x >= shift)
			return 1.0f + (x - shift) * 0.01;
		float tmp = (x - shift) * rshift;
		return 1.0f - tmp * tmp * tmp * tmp;
	} else if(x >= -shift) {
		float tmp = (x + shift) * rshift;
		return -1.0f + tmp * tmp * tmp * tmp;
	} else {
		return -1.0f - (shift - x) * 0.01;
	}
}

__device__ float NeuralNetwork::DeviceEvaluator::Evaluate(const float *features) const {
	const int MAX_LAYER_SIZE = maxLayerSize;
	float activations[MAX_LAYER_SIZE];

	// if (this->max_layer_size <= 0 || this->max_layer_size > 101 /*Match static const*/) { // Basic sanity check
    //     return 0.0f; // Or handle error differently if critical path allows
    // }
	if(threadIdx.x == 0 && blockIdx.x == 0) {
		for(int i = 0; i < this->d_topology[0]; i++) {
			if(isnan(features[i]) || isinf(features[i])) {
				printf("[ERROR] Invalid input feature at index %d: %f\n", i, features[i]);
				NeuralNetwork::DeviceEvaluator::ReportAndAbort("Invalid input feature");
				return 0.0f;
			}
		}
	}
	
	// 2. Copy input (without printing)
	for(int i = 0; i < this->d_topology[0]; i++) {
		activations[i] = features[i];
	}

	int weight_offset = 0;
	int bias_offset = 0;

	// Calculate totals without printing
	int total_weights_for_eval = 0;
	int total_biases_for_eval = 0;
	for(int i = 1; i < this->num_layers; i++) {
		total_weights_for_eval += this->d_topology[i - 1] * this->d_topology[i];
		total_biases_for_eval += this->d_topology[i];
	}

	for(int layer = 1; layer < this->num_layers; layer++) {
		int in_size = this->d_topology[layer - 1];
		int out_size = this->d_topology[layer];


		for(int neuron = 0; neuron < out_size; neuron++) {
			float sum = this->biases[bias_offset + neuron];

			for(int i = 0; i < in_size; i++) {
				int weight_idx = weight_offset + neuron * in_size + i;

				sum += activations[i] * this->weights[weight_idx];
			}

			activations[neuron] = ScaleTanh2(sum);
		}

		weight_offset += in_size * out_size;
		bias_offset += out_size;
	}

	float final_output = (this->d_topology[this->num_layers - 1] == 1) ? activations[0] : 0.0f;

	return final_output;
}

// New Evaluate function using shared memory pointers
__device__ float NeuralNetwork::DeviceEvaluator::Evaluate(const float* features, const float* p_shared_weights, const float* p_shared_biases) const {
    // Use this->max_layer_size which is now set correctly

    if (this->max_layer_size <= 0 || this->max_layer_size > 101 /*Match static const*/) { // Basic sanity check
        return 0.0f; // Or handle error differently if critical path allows
    }
    // Using 101 directly as per existing code's use of NeuralNetwork::maxLayerSize
    float activations[maxLayerSize]; // Max size for activations array on stack

	if(threadIdx.x == 0 && blockIdx.x == 0) {
		for(int i = 0; i < this->d_topology[0]; i++) {
			if(isnan(features[i]) || isinf(features[i])) {
				printf("[ERROR] Invalid input feature at index %d: %f\n", i, features[i]);
				NeuralNetwork::DeviceEvaluator::ReportAndAbort("Invalid input feature");
				return 0.0f;
			}
		}
	}

    // Input features copy (checks removed for performance as per your successful test)
    for(int i = 0; i < this->d_topology[0]; i++) {
        activations[i] = features[i];
    }

    int weight_idx_offset = 0; // Offset for reading from p_shared_weights
    int bias_idx_offset = 0;   // Offset for reading from p_shared_biases

    for(int layer = 1; layer < this->num_layers; layer++) {
        int in_size = this->d_topology[layer - 1];
        int out_size = this->d_topology[layer];
        
        float next_activations[101]; // Temporary buffer for next layer's activations

        for(int neuron = 0; neuron < out_size; neuron++) {
            float sum = p_shared_biases[bias_idx_offset + neuron]; // Read from shared biases

            for(int i = 0; i < in_size; i++) {
                // Read from shared weights
                float weight_val = p_shared_weights[weight_idx_offset + neuron * in_size + i];
                sum += activations[i] * weight_val;
            }
            next_activations[neuron] = ScaleTanh2(sum); 
        }
        

        // for(int i=0; i < out_size; ++i) { // Copy to activations for next layer
        //     activations[i] = next_activations[i];
        // }

		memcpy(activations, next_activations, out_size * sizeof(float));

        weight_idx_offset += in_size * out_size;
        bias_idx_offset += out_size;
    }

    // Assuming output layer has 1 neuron for FJSS evaluation score
    float final_output = (this->d_topology[this->num_layers - 1] == 1) ? activations[0] : 0.0f;
    return final_output;
}


void NeuralNetwork::GenerateWeights() {
	weights.clear();
	for(size_t i = 1; i < topology.size(); ++i) {
		std::vector<float> layerWeights(topology[i] * topology[i - 1]);
		float range = sqrt(6.0f / (topology[i - 1] + topology[i]));	 // Xavier initialization
		for(float &weight: layerWeights) {
			weight = (rand() / (float)RAND_MAX) * 2 * range - range;
		}
		this->weights.push_back(layerWeights);
	}
	// }
}

void NeuralNetwork::GenerateBiases() {
	biases.clear();
	for(size_t i = 1; i < topology.size(); ++i) {
		std::vector<float> layerBiases(topology[i], 0.1f);
		this->biases.push_back(layerBiases);
	}
}

void NeuralNetwork::FlattenParams() {
	flattenedWeights.clear();
	flattenedBiases.clear();

	for(const auto &layer: weights) {
		flattenedWeights.insert(flattenedWeights.end(),
								layer.begin(), layer.end());
	}

	for(const auto &layer: biases) {
		flattenedBiases.insert(flattenedBiases.end(),
							   layer.begin(), layer.end());
	}
}

std::vector<NeuralNetwork> NeuralNetwork::LoadBatchFromJson(const std::string &filename) {
	std::string full_path = FileManager::GetFullPath(filename);
	std::ifstream in(full_path);
	if(!in) throw std::runtime_error("Cannot open weights file: " + full_path);

	nlohmann::json all_nets;
	in >> all_nets;
	in.close();

	std::vector<NeuralNetwork> networks;
	for(const auto &j: all_nets) {
		std::vector<int> topology = j["topology"].get<std::vector<int>>();
		std::vector<std::vector<float>> weights = j["weights"].get<std::vector<std::vector<float>>>();
		std::vector<std::vector<float>> biases = j["biases"].get<std::vector<std::vector<float>>>();
		networks.emplace_back(topology, &weights, &biases);
	}
	return networks;
}