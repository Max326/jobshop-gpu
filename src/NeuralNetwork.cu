#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cmath>
#include <cstdlib>
#include <ctime>
#include <fstream>

#include "NeuralNetwork.h"

struct NeuralNetwork::CudaData {
	float *d_weights = nullptr;
	float *d_biases = nullptr;
	float *d_input = nullptr;
	float *d_output = nullptr;
};

// Konstruktor przenoszący
NeuralNetwork::NeuralNetwork(NeuralNetwork &&other) noexcept
	: topology(std::move(other.topology)),
	  weights(std::move(other.weights)),
	  biases(std::move(other.biases)),
	  layerOffsets(std::move(other.layerOffsets)),
	  biasOffsets(std::move(other.biasOffsets)),
	  cudaData(std::move(other.cudaData)) {
	// Zabezpieczenie przed podwójnym zwolnieniem pamięci
	other.cudaData.reset(nullptr);
}

// Operator przypisania przenoszącego
NeuralNetwork &NeuralNetwork::operator=(NeuralNetwork &&other) noexcept {
	if(this != &other) {
		topology = std::move(other.topology);
		weights = std::move(other.weights);
		biases = std::move(other.biases);
		layerOffsets = std::move(other.layerOffsets);
		biasOffsets = std::move(other.biasOffsets);
		cudaData = std::move(other.cudaData);
	}
	return *this;
}

// Funkcja aktywacji scaleTanh2
__device__ float ScaleTanh2(float x) {
	constexpr float shift = 3.5f;
	constexpr float rshift = 1.0f / shift;
	if(x >= 0.f) {
		if(x >= shift)
			return 1.0f + (x - shift) * 0.01;
		float tmp = (x - shift) * rshift;
		return 1.0f - tmp * tmp * tmp * tmp;
	} else if(x >= -shift) {
		float tmp = (x + shift) * rshift;
		return -1.0f + tmp * tmp * tmp * tmp;
	} else {
		return -1.0f - (shift - x) * 0.01;
	}
}


// Example optimized kernel using shared memory
__global__ void ForwardPassKernel(const float *input, int inputSize,
								  const float *weights, const float *biases,
								  float *output, int outputSize) {
	extern __shared__ float shared_input[];
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// Load input into shared memory (coalesced access)
	if(threadIdx.x < inputSize) {
		shared_input[threadIdx.x] = input[threadIdx.x];
	}
	__syncthreads();

	if(idx < outputSize) {
		float sum = 0.0f;
		for(int i = 0; i < inputSize; ++i) {
			sum += shared_input[i] * weights[idx * inputSize + i];
		}
		sum += biases[idx];
		output[idx] = ScaleTanh2(sum);
	}
}

NeuralNetwork::NeuralNetwork(const std::vector<int> &topology,
							 const std::vector<std::vector<float>> *weights_ptr,
							 const std::vector<std::vector<float>> *biases_ptr)
	: topology(topology),
	  weights(weights_ptr ? *weights_ptr : std::vector<std::vector<float>>()),
	  biases(biases_ptr ? *biases_ptr : std::vector<std::vector<float>>()),
	  cudaData(std::make_unique<CudaData>()) {
	// ======================
	//  Validation checks
	// ======================

	if(weights_ptr == nullptr) {
		GenerateWeights();
	}
	if(biases_ptr == nullptr) {
		GenerateBiases();
	}

	// 1. Validate topology
	if(topology.empty()) {
		throw std::invalid_argument("NeuralNetwork: Topology cannot be empty");
	}

	if(topology.size() < 2) {
		throw std::invalid_argument("NeuralNetwork: Topology must have at least 2 layers (input/output)");
	}

	// 2. Validate weights/biases structure
	const size_t num_weight_layers = topology.size() - 1;

	if(weights.size() != num_weight_layers) {
		throw std::invalid_argument("NeuralNetwork: Incorrect number of weight matrices. Expected " +
									std::to_string(num_weight_layers) + ", got " +
									std::to_string(weights.size()));
	}

	if(biases.size() != num_weight_layers) {
		throw std::invalid_argument("NeuralNetwork: Incorrect number of bias vectors. Expected " +
									std::to_string(num_weight_layers) + ", got " +
									std::to_string(biases.size()));
	}

	// 3. Validate individual layer dimensions
	for(size_t i = 0; i < weights.size(); ++i) {
		const int expected_weights = topology[i] * topology[i + 1];
		if(weights[i].size() != static_cast<size_t>(expected_weights)) {
			throw std::invalid_argument("NeuralNetwork: Weights matrix at layer " +
										std::to_string(i) + " has incorrect size. Expected " +
										std::to_string(expected_weights) + ", got " +
										std::to_string(weights[i].size()));
		}
	}

	for(size_t i = 0; i < biases.size(); ++i) {
		const int expected_biases = topology[i + 1];
		if(biases[i].size() != static_cast<size_t>(expected_biases)) {
			throw std::invalid_argument("NeuralNetwork: Bias vector at layer " +
										std::to_string(i) + " has incorrect size. Expected " +
										std::to_string(expected_biases) + ", got " +
										std::to_string(biases[i].size()));
		}
	}

	// 1. Calculate offsets for each layer's weights and biases
	layerOffsets.resize(this->weights.size());
	biasOffsets.resize(biases.size());
	size_t total_weights = 0;
	size_t total_biases = 0;

	for(size_t i = 0; i < this->weights.size(); ++i) {
		layerOffsets[i] = total_weights;
		total_weights += this->weights[i].size();

		biasOffsets[i] = total_biases;
		total_biases += this->biases[i].size();
	}

	// 2. Allocate GPU memory for weights and biases
	CUDA_CHECK(hipMalloc(&cudaData->d_weights, total_weights * sizeof(float)));
	CUDA_CHECK(hipMalloc(&cudaData->d_biases, total_biases * sizeof(float)));

	// 3. Find the maximum layer size for input/output buffers
	int max_layer_size = 0;
	for(int size: topology) {
		if(size > max_layer_size) {
			max_layer_size = size;
		}
	}

	// Allocate input and output buffers to the maximum layer size
	CUDA_CHECK(hipMalloc(&cudaData->d_input, max_layer_size * sizeof(float)));
	CUDA_CHECK(hipMalloc(&cudaData->d_output, max_layer_size * sizeof(float)));

	// 4. Copy weights and biases to GPU
	size_t weight_offset = 0;
	size_t bias_offset = 0;

	for(size_t i = 0; i < this->weights.size(); ++i) {
		CUDA_CHECK(hipMemcpy(cudaData->d_weights + weight_offset,
							  this->weights[i].data(),
							  this->weights[i].size() * sizeof(float),
							  hipMemcpyHostToDevice));

		CUDA_CHECK(hipMemcpy(cudaData->d_biases + bias_offset,
							  this->biases[i].data(),
							  this->biases[i].size() * sizeof(float),
							  hipMemcpyHostToDevice));

		weight_offset += this->weights[i].size();
		bias_offset += this->biases[i].size();
	}

	// W konstruktorze, po załadowaniu wag:
	std::cout << "=== DATA VALIDATION ===\n";
	std::cout << "Topology: ";
	for(auto t: topology)
		std::cout << t << " ";
	std::cout << "\nFirst weight layer: " << weights[0][0] << ", " << weights[0][1] << "...\n";
	std::cout << "First bias: " << biases[0][0] << "\n";
}

NeuralNetwork::~NeuralNetwork() {
	if(cudaData) {
		hipFree(cudaData->d_weights);
		hipFree(cudaData->d_biases);
		hipFree(cudaData->d_input);
		hipFree(cudaData->d_output);
	}
}
std::vector<float> NeuralNetwork::Forward(const std::vector<float> &input) {
	// Copy input to device
	CUDA_CHECK(hipMemcpy(cudaData->d_input, input.data(),
						  input.size() * sizeof(float),
						  hipMemcpyHostToDevice));

	float *current_input = cudaData->d_input;
	float *current_output = cudaData->d_output;

	for(size_t l = 0; l < weights.size(); ++l) {
		int in_size = topology[l];
		int out_size = topology[l + 1];

		// Get the weight and bias offsets for this layer
		size_t weight_offset = layerOffsets[l];
		size_t bias_offset = biasOffsets[l];

		// Launch the kernel
		int threadsPerBlock = 256;
		int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;

		size_t sharedMemSize = in_size * sizeof(float); // in_size = current input layer size

		ForwardPassKernel<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(
			current_input, in_size,
			cudaData->d_weights + weight_offset,
			cudaData->d_biases + bias_offset,
			current_output, out_size);
		CUDA_CHECK(hipGetLastError());
		CUDA_CHECK(hipDeviceSynchronize());

		// Swap input and output buffers if not the last layer
		if(l != weights.size() - 1) {
			std::swap(current_input, current_output);
		}
	}

	// Copy the final output from device to host
	std::vector<float> output(topology.back());
	CUDA_CHECK(hipMemcpy(output.data(), current_output,
						  topology.back() * sizeof(float),
						  hipMemcpyDeviceToHost));

	return output;
}

void NeuralNetwork::GenerateWeights() {
	weights.clear();
	// if(weights != nullptr) {
	// this->weights = *weights;
	// } else {
	for(size_t i = 1; i < topology.size(); ++i) {
		std::vector<float> layerWeights(topology[i] * topology[i - 1]);
		float range = sqrt(6.0f / (topology[i - 1] + topology[i]));	 // Xavier initialization
		for(float &weight: layerWeights) {
			weight = (rand() / (float)RAND_MAX) * 2 * range - range;
		}
		this->weights.push_back(layerWeights);
	}
	// }
}

void NeuralNetwork::GenerateBiases() {
	biases.clear();
	for(size_t i = 1; i < topology.size(); ++i) {
		std::vector<float> layerBiases(topology[i], 0.1f);
		this->biases.push_back(layerBiases);
	}
}
