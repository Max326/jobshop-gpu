#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cmath>
#include <cstdlib>
#include <ctime>
#include <fstream>

#include "NeuralNetwork.cuh"


void NeuralNetwork::InitializeCudaData() {
	// 1. Calculate offsets for each layer's weights and biases
	FlattenParams();

	std::cout << "Flattened weights size: " << flattenedWeights.size() << "\n";
	std::cout << "Flattened biases size: " << flattenedBiases.size() << "\n";
	std::cout << "First few weights: ";
	for(int i = 0; i < std::min(5, (int)flattenedWeights.size()); i++)
		std::cout << flattenedWeights[i] << " ";
	std::cout << "\n";

	layerOffsets.resize(weights.size());
	biasOffsets.resize(biases.size());
	size_t total_weights = 0;
	size_t total_biases = 0;

	for(size_t i = 0; i < weights.size(); ++i) {
		layerOffsets[i] = total_weights;
		total_weights += weights[i].size();

		biasOffsets[i] = total_biases;
		total_biases += biases[i].size();
	}

	// 2. Allocate GPU memory for weights and biases
	CUDA_CHECK(hipMalloc(&cudaData->d_weights, total_weights * sizeof(float)));
	CUDA_CHECK(hipMalloc(&cudaData->d_biases, total_biases * sizeof(float)));

	// 3. Find the maximum layer size for input/output buffers
	int max_layer_size = 0;
	for(int size: topology) {
		if(size > max_layer_size) {
			max_layer_size = size;
		}
	}

	// Allocate input and output buffers to the maximum layer size
	CUDA_CHECK(hipMalloc(&cudaData->d_input, max_layer_size * sizeof(float)));
	CUDA_CHECK(hipMalloc(&cudaData->d_output, max_layer_size * sizeof(float)));

	// 4. Copy weights and biases to GPU
	size_t weight_offset = 0;
	size_t bias_offset = 0;

	for(size_t i = 0; i < weights.size(); ++i) {
		CUDA_CHECK(hipMemcpy(cudaData->d_weights + weight_offset,
							  weights[i].data(),
							  weights[i].size() * sizeof(float),
							  hipMemcpyHostToDevice));

		CUDA_CHECK(hipMemcpy(cudaData->d_biases + bias_offset,
							  biases[i].data(),
							  biases[i].size() * sizeof(float),
							  hipMemcpyHostToDevice));

		weight_offset += weights[i].size();
		bias_offset += biases[i].size();
	}
}

NeuralNetwork::NeuralNetwork() : cudaData(std::make_unique<CudaData>()) {}

NeuralNetwork::NeuralNetwork(const std::vector<int> &topology,
							 const std::vector<std::vector<float>> *weights_ptr,
							 const std::vector<std::vector<float>> *biases_ptr)
	: topology(topology),
	  weights(weights_ptr ? *weights_ptr : std::vector<std::vector<float>>()),
	  biases(biases_ptr ? *biases_ptr : std::vector<std::vector<float>>()),
	  cudaData(std::make_unique<CudaData>()) {
	// ======================
	//  Validation checks
	// ======================

	if(weights_ptr == nullptr) {
		GenerateWeights();
	}
	if(biases_ptr == nullptr) {
		GenerateBiases();
	}

	// 1. Validate topology
	if(topology.empty()) {
		throw std::invalid_argument("NeuralNetwork: Topology cannot be empty");
	}

	if(topology.size() < 2) {
		throw std::invalid_argument("NeuralNetwork: Topology must have at least 2 layers (input/output)");
	}

	Validate();
	InitializeCudaData();
}

NeuralNetwork::~NeuralNetwork() {
	if(cudaData) {
		hipFree(cudaData->d_weights);
		hipFree(cudaData->d_biases);
		hipFree(cudaData->d_input);
		hipFree(cudaData->d_output);
	}
}

// Konstruktor przenoszący
NeuralNetwork::NeuralNetwork(NeuralNetwork &&other) noexcept
	: topology(std::move(other.topology)),
	  weights(std::move(other.weights)),
	  biases(std::move(other.biases)),
	  layerOffsets(std::move(other.layerOffsets)),
	  biasOffsets(std::move(other.biasOffsets)),
	  cudaData(std::move(other.cudaData)) {
	// Zabezpieczenie przed podwójnym zwolnieniem pamięci
	other.cudaData.reset(nullptr);
}

// Operator przypisania przenoszącego
NeuralNetwork &NeuralNetwork::operator=(NeuralNetwork &&other) noexcept {
	if(this != &other) {
		topology = std::move(other.topology);
		weights = std::move(other.weights);
		biases = std::move(other.biases);
		layerOffsets = std::move(other.layerOffsets);
		biasOffsets = std::move(other.biasOffsets);
		cudaData = std::move(other.cudaData);
	}
	return *this;
}

// Funkcja aktywacji scaleTanh2
__device__ float ScaleTanh2(float x) {
	constexpr float shift = 3.5f;
	constexpr float rshift = 1.0f / shift;
	if(x >= 0.f) {
		if(x >= shift)
			return 1.0f + (x - shift) * 0.01;
		float tmp = (x - shift) * rshift;
		return 1.0f - tmp * tmp * tmp * tmp;
	} else if(x >= -shift) {
		float tmp = (x + shift) * rshift;
		return -1.0f + tmp * tmp * tmp * tmp;
	} else {
		return -1.0f - (shift - x) * 0.01;
	}
}

__device__ float NeuralNetwork::DeviceEvaluator::Evaluate(const float *features) const {
	const int MAX_LAYER_SIZE = 32;	// Match your header definition
	float activations[MAX_LAYER_SIZE];

	// 1. Validate input size
	if(topology[0] > MAX_LAYER_SIZE) return 0.0f;

	// 2. Copy input with bounds checking
	for(int i = 0; i < topology[0] && i < MAX_LAYER_SIZE; i++) {
		activations[i] = features[i];
	}

	int weight_offset = 0;
	int bias_offset = 0;
	int total_weights = 0;
	int total_biases = 0;

	// 3. Pre-calculate total weights/biases for bounds checking
	for(int i = 1; i < num_layers; i++) {
		total_weights += topology[i - 1] * topology[i];
		total_biases += topology[i];
	}

	for(int layer = 1; layer < num_layers; layer++) {
		int in_size = topology[layer - 1];
		int out_size = topology[layer];

		// 4. Validate layer dimensions
		if(out_size > MAX_LAYER_SIZE) return 0.0f;

		for(int neuron = 0; neuron < out_size; neuron++) {
			// 5. Check bias access
			if(bias_offset + neuron >= total_biases) {
				printf("Bias access out of bounds: %d >= %d\n",
					   bias_offset + neuron, total_biases);
				return 0.0f;
			}

			float sum = biases[bias_offset + neuron];

			for(int i = 0; i < in_size; i++) {
				// 6. Check weight access
				int weight_idx = weight_offset + neuron * in_size + i;
				if(weight_idx >= total_weights) {
					printf("Weight access out of bounds: %d >= %d\n",
						   weight_idx, total_weights);
					return 0.0f;
				}

				sum += activations[i] * weights[weight_idx];
			}

			activations[neuron] = ScaleTanh2(sum);
		}

		weight_offset += in_size * out_size;
		bias_offset += out_size;
	}

	return activations[0];
}

void NeuralNetwork::GenerateWeights() {
	weights.clear();
	for(size_t i = 1; i < topology.size(); ++i) {
		std::vector<float> layerWeights(topology[i] * topology[i - 1]);
		float range = sqrt(6.0f / (topology[i - 1] + topology[i]));	 // Xavier initialization
		for(float &weight: layerWeights) {
			weight = (rand() / (float)RAND_MAX) * 2 * range - range;
		}
		this->weights.push_back(layerWeights);
	}
	// }
}

void NeuralNetwork::GenerateBiases() {
	biases.clear();
	for(size_t i = 1; i < topology.size(); ++i) {
		std::vector<float> layerBiases(topology[i], 0.1f);
		this->biases.push_back(layerBiases);
	}
}

void NeuralNetwork::FlattenParams() {
	flattenedWeights.clear();
	flattenedBiases.clear();

	for(const auto &layer: weights) {
		flattenedWeights.insert(flattenedWeights.end(),
								layer.begin(), layer.end());
	}

	for(const auto &layer: biases) {
		flattenedBiases.insert(flattenedBiases.end(),
							   layer.begin(), layer.end());
	}
}