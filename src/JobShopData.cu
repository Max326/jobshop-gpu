#include <hip/hip_runtime.h>

#include "JobShopData.cuh"

GPUProblem JobShopDataGPU::UploadToGPU(const JobShopData& problem) {
	GPUProblem gpuProblem;

	// 1. Copy basic info
	gpuProblem.numMachines = problem.numMachines;
	gpuProblem.numJobs = problem.numJobs;
	gpuProblem.numOpTypes = problem.numOpTypes;

	// 2. Process jobs (using pinned memory for better performance)
	std::vector<GPUJob> hostJobs(problem.numJobs);
	hipMalloc(&gpuProblem.jobs, sizeof(GPUJob) * problem.numJobs);

	for(int j = 0; j < problem.numJobs; j++) {
		const auto& cpuJob = problem.jobs[j];
		GPUJob& gpuJob = hostJobs[j];

		gpuJob.id = cpuJob.id;
		// gpuJob.nextOpIndex = cpuJob.nextOpIndex;
		// gpuJob.lastOpEndTime = cpuJob.lastOpEndTime;
		gpuJob.operationCount = cpuJob.operations.size();

		// Allocate operations
		hipMalloc(&gpuJob.operations, sizeof(GPUOperation) * gpuJob.operationCount);
		std::vector<GPUOperation> hostOps(gpuJob.operationCount);

		for(int o = 0; o < gpuJob.operationCount; o++) {
			const auto& cpuOp = cpuJob.operations[o];
			GPUOperation& gpuOp = hostOps[o];

			gpuOp.type = cpuOp.type;
			gpuOp.eligibleCount = cpuOp.eligibleMachines.size();

			// Allocate and copy eligible machines
			hipMalloc(&gpuOp.eligibleMachines, sizeof(int) * gpuOp.eligibleCount);
			hipMemcpy(gpuOp.eligibleMachines, cpuOp.eligibleMachines.data(),
					   sizeof(int) * gpuOp.eligibleCount, hipMemcpyHostToDevice);
		}

		// Copy operations to device
		hipMemcpy(gpuJob.operations, hostOps.data(),
				   sizeof(GPUOperation) * gpuJob.operationCount, hipMemcpyHostToDevice);
	}

	// CRITICAL: Copy jobs array to device
	hipMemcpy(gpuProblem.jobs, hostJobs.data(),
			   sizeof(GPUJob) * problem.numJobs, hipMemcpyHostToDevice);

	// 3. Process processing times
	std::vector<int> flatTimes(problem.numOpTypes * problem.numMachines);
	for(int o = 0; o < problem.numOpTypes; o++) {
		for(int m = 0; m < problem.numMachines; m++) {
			flatTimes[o * problem.numMachines + m] = problem.processingTimes[o][m];
		}
	}
	hipMalloc(&gpuProblem.processingTimes, sizeof(int) * flatTimes.size());
	hipMemcpy(gpuProblem.processingTimes, flatTimes.data(),
			   sizeof(int) * flatTimes.size(), hipMemcpyHostToDevice);

	// Error checking
	hipError_t err = hipGetLastError();
	if(err != hipSuccess) {
		FreeGPUData(gpuProblem);  // Clean up if error
		throw std::runtime_error("CUDA error during upload: " +
								 std::string(hipGetErrorString(err)));
	}

	return gpuProblem;
}

void JobShopDataGPU::FreeGPUData(GPUProblem& gpuProblem) {
	// Helper function to free nested structures
	auto FreeJob = [](GPUJob& job) {
		if(job.operations) {
			std::vector<GPUOperation> tempOps(job.operationCount);
			hipMemcpy(tempOps.data(), job.operations,
					   sizeof(GPUOperation) * job.operationCount, hipMemcpyDeviceToHost);

			for(auto& op: tempOps) {
				if(op.eligibleMachines) {
					hipFree(op.eligibleMachines);
				}
			}
			hipFree(job.operations);
		}
	};

	// 1. Free jobs and their nested data
	if(gpuProblem.jobs) {
		std::vector<GPUJob> tempJobs(gpuProblem.numJobs);
		hipMemcpy(tempJobs.data(), gpuProblem.jobs,
				   sizeof(GPUJob) * gpuProblem.numJobs, hipMemcpyDeviceToHost);

		for(auto& job: tempJobs) {
			FreeJob(job);
		}
		hipFree(gpuProblem.jobs);
	}

	// 2. Free processing times
	if(gpuProblem.processingTimes) {
		hipFree(gpuProblem.processingTimes);
	}

	// 3. Reset struct
	gpuProblem = GPUProblem {};
}