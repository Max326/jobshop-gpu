#include <hip/hip_runtime.h>

#include "JobShopData.cuh"

GPUProblem JobShopDataGPU::UploadToGPU(const JobShopData& problem) {
	GPUProblem gpuProblem;

	// 1. Copy basic info
	gpuProblem.numMachines = problem.numMachines;
	gpuProblem.numJobs = problem.numJobs;
	gpuProblem.numOpTypes = problem.numOpTypes;

	// 2. Process jobs (using pinned memory for better performance)
	std::vector<GPUJob> hostJobs(problem.numJobs);
	hipMalloc(&gpuProblem.jobs, sizeof(GPUJob) * problem.numJobs);

	for(int j = 0; j < problem.numJobs; j++) {
		const auto& cpuJob = problem.jobs[j];
		GPUJob& gpuJob = hostJobs[j];

		gpuJob.id = cpuJob.id;
		// gpuJob.nextOpIndex = cpuJob.nextOpIndex;
		// gpuJob.lastOpEndTime = cpuJob.lastOpEndTime;
		gpuJob.operationCount = cpuJob.operations.size();

		// Allocate operations
		hipMalloc(&gpuJob.operations, sizeof(GPUOperation) * gpuJob.operationCount);
		std::vector<GPUOperation> hostOps(gpuJob.operationCount);

		for(int o = 0; o < gpuJob.operationCount; o++) {
			const auto& cpuOp = cpuJob.operations[o];
			GPUOperation& gpuOp = hostOps[o];

			gpuOp.type = cpuOp.type;
			gpuOp.predecessorCount = cpuOp.predecessorCount;
			gpuOp.lastPredecessorEndTime = cpuOp.lastPredecessorEndTime;

			gpuOp.eligibleCount = cpuOp.eligibleMachines.size();

			// Allocate and copy eligible machines
			hipMalloc(&gpuOp.eligibleMachines, sizeof(int) * gpuOp.eligibleCount);
			hipMemcpy(gpuOp.eligibleMachines, cpuOp.eligibleMachines.data(),
					   sizeof(int) * gpuOp.eligibleCount, hipMemcpyHostToDevice);

			gpuOp.successorCount = cpuOp.successorsIDs.size();
			hipMalloc(&gpuOp.successorsIDs, sizeof(int) * gpuOp.successorCount);
			hipMemcpy(gpuOp.successorsIDs, cpuOp.successorsIDs.data(),
					   sizeof(int) * gpuOp.successorCount, hipMemcpyHostToDevice);
		}

		// Copy operations to device
		hipMemcpy(gpuJob.operations, hostOps.data(),
				   sizeof(GPUOperation) * gpuJob.operationCount, hipMemcpyHostToDevice);
	}

	// CRITICAL: Copy jobs array to device
	hipMemcpy(gpuProblem.jobs, hostJobs.data(),
			   sizeof(GPUJob) * problem.numJobs, hipMemcpyHostToDevice);

	// 3. Process processing times
	std::vector<int> flatTimes(problem.numOpTypes * problem.numMachines);
	for(int o = 0; o < problem.numOpTypes; o++) {
		for(int m = 0; m < problem.numMachines; m++) {
			flatTimes[o * problem.numMachines + m] = problem.processingTimes[o][m];
		}
	}
	hipMalloc(&gpuProblem.processingTimes, sizeof(int) * flatTimes.size());
	hipMemcpy(gpuProblem.processingTimes, flatTimes.data(),
			   sizeof(int) * flatTimes.size(), hipMemcpyHostToDevice);

	// Error checking
	hipError_t err = hipGetLastError();
	if(err != hipSuccess) {
		FreeGPUData(gpuProblem);  // Clean up if error
		throw std::runtime_error("CUDA error during upload: " +
								 std::string(hipGetErrorString(err)));
	}

	return gpuProblem;
}

GPUProblem JobShopDataGPU::UploadParallelToGPU(const JobShopData& problem) {
    GPUProblem gpuProblem;
    
    // 1. Copy basic dimensions
    gpuProblem.numMachines = problem.numMachines;
    gpuProblem.numJobs = problem.numJobs;
    gpuProblem.numOpTypes = problem.numOpTypes;

    // 2. Upload processing times
    std::vector<int> flatTimes(problem.numOpTypes * problem.numMachines, 0);
    for (int t = 0; t < problem.numOpTypes; ++t) {
        for (int m = 0; m < problem.numMachines; ++m) {
            flatTimes[t * problem.numMachines + m] = problem.processingTimes[t][m];
        }
    }
    hipMalloc(&gpuProblem.processingTimes, flatTimes.size() * sizeof(int));
    hipMemcpy(gpuProblem.processingTimes, flatTimes.data(), 
              flatTimes.size() * sizeof(int), hipMemcpyHostToDevice);

    // 3. Upload jobs and operations
    std::vector<GPUJob> hostJobs(problem.numJobs);
    hipMalloc(&gpuProblem.jobs, problem.numJobs * sizeof(GPUJob));
    
    for (int j = 0; j < problem.numJobs; ++j) {
        const auto& cpuJob = problem.jobs[j];
        GPUJob gpuJob;
        gpuJob.id = cpuJob.id;
        gpuJob.operationCount = cpuJob.operations.size();

        // Upload operations
        std::vector<GPUOperation> hostOps(cpuJob.operations.size());
        hipMalloc(&gpuJob.operations, hostOps.size() * sizeof(GPUOperation));
        
        for (size_t o = 0; o < cpuJob.operations.size(); ++o) {
            const auto& cpuOp = cpuJob.operations[o];
            GPUOperation gpuOp;
            
            // Copy operation metadata
            gpuOp.type = cpuOp.type;
            gpuOp.predecessorCount = cpuOp.predecessorCount;
            gpuOp.lastPredecessorEndTime = cpuOp.lastPredecessorEndTime;

            // Upload eligible machines
            hipMalloc(&gpuOp.eligibleMachines, cpuOp.eligibleMachines.size() * sizeof(int));
            hipMemcpy(gpuOp.eligibleMachines, cpuOp.eligibleMachines.data(),
                      cpuOp.eligibleMachines.size() * sizeof(int), hipMemcpyHostToDevice);
            gpuOp.eligibleCount = cpuOp.eligibleMachines.size();

            // Upload successors
            hipMalloc(&gpuOp.successorsIDs, cpuOp.successorsIDs.size() * sizeof(int));
            hipMemcpy(gpuOp.successorsIDs, cpuOp.successorsIDs.data(),
                      cpuOp.successorsIDs.size() * sizeof(int), hipMemcpyHostToDevice);
            gpuOp.successorCount = cpuOp.successorsIDs.size();

            hostOps[o] = gpuOp;
        }
        
        hipMemcpy(gpuJob.operations, hostOps.data(),
                  hostOps.size() * sizeof(GPUOperation), hipMemcpyHostToDevice);
        hostJobs[j] = gpuJob;
    }

    hipMemcpy(gpuProblem.jobs, hostJobs.data(),
              hostJobs.size() * sizeof(GPUJob), hipMemcpyHostToDevice);

    return gpuProblem;
}

void JobShopDataGPU::FreeGPUData(GPUProblem& gpuProblem) {
	// Helper function to free nested structures
	auto FreeJob = [](GPUJob& job) {
		if(job.operations) {
			std::vector<GPUOperation> tempOps(job.operationCount);
			hipMemcpy(tempOps.data(), job.operations,
					   sizeof(GPUOperation) * job.operationCount, hipMemcpyDeviceToHost);

			for(auto& op: tempOps) {
				if(op.eligibleMachines) {
					hipFree(op.eligibleMachines);
					hipFree(op.successorsIDs);
				}
			}
			hipFree(job.operations);
		}
	};

	// 1. Free jobs and their nested data
	if(gpuProblem.jobs) {
		std::vector<GPUJob> tempJobs(gpuProblem.numJobs);
		hipMemcpy(tempJobs.data(), gpuProblem.jobs,
				   sizeof(GPUJob) * gpuProblem.numJobs, hipMemcpyDeviceToHost);

		for(auto& job: tempJobs) {
			FreeJob(job);
		}
		hipFree(gpuProblem.jobs);
	}

	// 2. Free processing times
	if(gpuProblem.processingTimes) {
		hipFree(gpuProblem.processingTimes);
	}

	// 3. Reset struct
	gpuProblem = GPUProblem {};
}