#include <hip/hip_runtime.h>

#include "JobShopData.cuh"

GPUProblem JobShopDataGPU::UploadToGPU(const JobShopData& problem) {
    GPUProblem gpuProblem;

    // 1. Copy basic info
    gpuProblem.numMachines = problem.numMachines;
    gpuProblem.numJobs = problem.numJobs;
    gpuProblem.numOpTypes = problem.numOpTypes;

    // 2. count the ops, machines and succesors 
    int totalOps = 0, totalEligible = 0, totalSuccessors = 0;
    for (const auto& job : problem.jobs) {
        totalOps += job.operations.size();
        for (const auto& op : job.operations) {
            totalEligible += op.eligibleMachines.size();
            totalSuccessors += op.successorsIDs.size();
        }
    }

    // 3. Allocate
    std::vector<GPUJob> hostJobs(problem.numJobs);
    std::vector<GPUOperation> allOps(totalOps);
    std::vector<int> allEligible(totalEligible);
    std::vector<int> allSuccessors(totalSuccessors);

    // 4. fill 
    int opOffset = 0, eligibleOffset = 0, succOffset = 0;
    for (int j = 0; j < problem.numJobs; ++j) {
        const auto& cpuJob = problem.jobs[j];
        GPUJob& gpuJob = hostJobs[j];
        gpuJob.id = cpuJob.id;
        gpuJob.operationsOffset = opOffset;
        gpuJob.operationCount = cpuJob.operations.size();

        for (size_t o = 0; o < cpuJob.operations.size(); ++o) {
            const auto& cpuOp = cpuJob.operations[o];
            GPUOperation& gpuOp = allOps[opOffset];
            gpuOp.type = cpuOp.type;
            gpuOp.predecessorCount = cpuOp.predecessorCount;
            gpuOp.lastPredecessorEndTime = cpuOp.lastPredecessorEndTime;

            gpuOp.eligibleMachinesOffset = eligibleOffset;
            gpuOp.eligibleCount = cpuOp.eligibleMachines.size();
            for (size_t em = 0; em < cpuOp.eligibleMachines.size(); ++em)
                allEligible[eligibleOffset + em] = cpuOp.eligibleMachines[em];
            eligibleOffset += cpuOp.eligibleMachines.size();

            gpuOp.successorsOffset = succOffset;
            gpuOp.successorCount = cpuOp.successorsIDs.size();
            for (size_t s = 0; s < cpuOp.successorsIDs.size(); ++s)
                allSuccessors[succOffset + s] = cpuOp.successorsIDs[s];
            succOffset += cpuOp.successorsIDs.size();

            opOffset++;
        }
    }

    // 5. hipMalloc/hipMemcpy
    hipMalloc(&gpuProblem.jobs, sizeof(GPUJob) * problem.numJobs);
    hipMemcpy(gpuProblem.jobs, hostJobs.data(), sizeof(GPUJob) * problem.numJobs, hipMemcpyHostToDevice);

    hipMalloc(&gpuProblem.operations, sizeof(GPUOperation) * totalOps);
    hipMemcpy(gpuProblem.operations, allOps.data(), sizeof(GPUOperation) * totalOps, hipMemcpyHostToDevice);

    hipMalloc(&gpuProblem.eligibleMachines, sizeof(int) * totalEligible);
    hipMemcpy(gpuProblem.eligibleMachines, allEligible.data(), sizeof(int) * totalEligible, hipMemcpyHostToDevice);

    hipMalloc(&gpuProblem.successorsIDs, sizeof(int) * totalSuccessors);
    hipMemcpy(gpuProblem.successorsIDs, allSuccessors.data(), sizeof(int) * totalSuccessors, hipMemcpyHostToDevice);

    // 6. Processing times 
    std::vector<int> flatTimes(problem.numOpTypes * problem.numMachines);
    for(int o = 0; o < problem.numOpTypes; o++) {
        for(int m = 0; m < problem.numMachines; m++) {
            flatTimes[o * problem.numMachines + m] = problem.processingTimes[o][m];
        }
    }
    hipMalloc(&gpuProblem.processingTimes, sizeof(int) * flatTimes.size());
    hipMemcpy(gpuProblem.processingTimes, flatTimes.data(),
               sizeof(int) * flatTimes.size(), hipMemcpyHostToDevice);

    return gpuProblem;
}

GPUProblem JobShopDataGPU::UploadParallelToGPU(const JobShopData& problem) {
    return UploadToGPU(problem);
}

void JobShopDataGPU::FreeGPUData(GPUProblem& gpuProblem) {
    if(gpuProblem.jobs) hipFree(gpuProblem.jobs);
    if(gpuProblem.operations) hipFree(gpuProblem.operations);
    if(gpuProblem.eligibleMachines) hipFree(gpuProblem.eligibleMachines);
    if(gpuProblem.successorsIDs) hipFree(gpuProblem.successorsIDs);
    if(gpuProblem.processingTimes) hipFree(gpuProblem.processingTimes);
    gpuProblem = GPUProblem {};
}