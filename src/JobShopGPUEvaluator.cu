#include "hip/hip_runtime.h"
#include "JobShopGPUEvaluator.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>
#include <chrono>

JobShopGPUEvaluator::JobShopGPUEvaluator(
    const std::string& problem_file,
    const std::vector<int>& nn_topology,
    const int &population_size,
    const int problem_count,
    int problem_offset,
    int max_loaded_problems)
    : nn_topology_(nn_topology), nn_candidate_count_(population_size)
{
    d_ops_working_ = nullptr;
    current_d_ops_working_size_ = 0;

    // load part of problems
    int to_load = std::min(problem_count - problem_offset, max_loaded_problems);
    cpu_problems_ = JobShopData::LoadFromParallelJson(problem_file, to_load, problem_offset);
    if (cpu_problems_.empty())
        throw std::runtime_error("No problems loaded!");

    //* all problems at once 
    // cpu_problems_ = JobShopData::LoadFromParallelJson(problem_file, problem_count);//TODO fix nummber of problem assignment 
    // if (cpu_problems_.empty())
    //     throw std::runtime_error("No problems loaded!");

    
    max_ops_per_problem_ = 0;
    for (const auto& prob : cpu_problems_) {
        int ops = 0;
        for (const auto& job : prob.jobs)
            ops += job.operations.size();
        if (ops > max_ops_per_problem_) max_ops_per_problem_ = ops;
    }

    nn_total_params_ = NeuralNetwork::CalculateTotalParameters(nn_topology_);

    // Allocate and upload shared topology array
    if (!nn_topology_.empty()) {
        size_t topology_size_bytes = nn_topology_.size() * sizeof(int);
        CUDA_CHECK(hipMalloc(&d_shared_topology_array_, topology_size_bytes));
        CUDA_CHECK(hipMemcpy(d_shared_topology_array_, nn_topology_.data(), topology_size_bytes, hipMemcpyHostToDevice));
    } else {
        // Handle empty topology case if necessary, or ensure nn_topology_ is never empty
        d_shared_topology_array_ = nullptr;
    }

    nn_total_weights_per_network_ = 0;
    nn_total_biases_per_network_ = 0;
    for (size_t i = 1; i < nn_topology_.size(); ++i) {
        nn_total_weights_per_network_ += nn_topology_[i - 1] * nn_topology_[i];
        nn_total_biases_per_network_ += nn_topology_[i];
    }

    total_weights_size_ = (size_t)nn_candidate_count_ * nn_total_weights_per_network_ * sizeof(float);
    total_biases_size_ = (size_t)nn_candidate_count_ * nn_total_biases_per_network_ * sizeof(float);

    // Allocate pinned host memory
    CUDA_CHECK(hipHostAlloc(&h_pinned_all_weights_, total_weights_size_, hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc(&h_pinned_all_biases_, total_biases_size_, hipHostMallocDefault));

    // Allocate GPU memory
    CUDA_CHECK(hipMalloc(&d_all_candidate_weights_, total_weights_size_));
    CUDA_CHECK(hipMalloc(&d_all_candidate_biases_, total_biases_size_));

    // Create a temporary vector for initial DeviceEvaluator setup
    std::vector<NeuralNetwork::DeviceEvaluator> temp_host_evaluators(nn_candidate_count_);

    for (int r = 0; r < nn_candidate_count_; ++r) {
        temp_host_evaluators[r].weights = d_all_candidate_weights_ + (size_t)r * nn_total_weights_per_network_;
        temp_host_evaluators[r].biases  = d_all_candidate_biases_  + (size_t)r * nn_total_biases_per_network_;

        if (nn_topology_.size() > MAX_NN_LAYERS) { // MAX_NN_LAYERS is defined in NeuralNetwork.cuh
            throw std::runtime_error("JobShopGPUEvaluator: Network topology exceeds MAX_NN_LAYERS.");
        }
        for (size_t i = 0; i < nn_topology_.size(); ++i) {
            temp_host_evaluators[r].d_topology[i] = nn_topology_[i];
        }
        
        temp_host_evaluators[r].num_layers = nn_topology_.size();
        temp_host_evaluators[r].max_layer_size = NeuralNetwork::maxLayerSize; // Access static const member
    }

    // Allocate and copy DeviceEvaluators to GPU ONCE
    CUDA_CHECK(hipMalloc(&d_evaluators_, sizeof(NeuralNetwork::DeviceEvaluator) * nn_candidate_count_));
    CUDA_CHECK(hipMemcpy(d_evaluators_, temp_host_evaluators.data(), sizeof(NeuralNetwork::DeviceEvaluator) * nn_candidate_count_, hipMemcpyHostToDevice));
}

JobShopGPUEvaluator::JobShopGPUEvaluator(
    const std::string& problem_file,
    const std::vector<int>& nn_topology,
    const int &population_size,
    const int problem_count)
    : JobShopGPUEvaluator(problem_file, nn_topology, population_size, problem_count, 0, problem_count)
{}

JobShopGPUEvaluator::~JobShopGPUEvaluator() {
    FreeProblemDataGPU();
    hipFree(d_evaluators_);
    hipFree(d_all_candidate_weights_);
    hipFree(d_all_candidate_biases_);
    hipHostFree(h_pinned_all_weights_);  // Use hipHostFree for pinned memory
    hipHostFree(h_pinned_all_biases_);  // Use hipHostFree for pinned memory
    
    if (d_ops_working_ != nullptr) {
        CUDA_CHECK(hipFree(d_ops_working_));
        d_ops_working_ = nullptr;
    }

    if (d_shared_topology_array_ != nullptr) {
        CUDA_CHECK(hipFree(d_shared_topology_array_));
        d_shared_topology_array_ = nullptr;
    }
}

void JobShopGPUEvaluator::FreeProblemDataGPU() {
    JobShopDataGPU::FreeBatchGPUData(d_problems_, d_jobs_, d_ops_, d_eligible_, d_succ_, d_procTimes_);
    d_problems_ = nullptr;
    d_jobs_ = nullptr;
    d_ops_ = nullptr;
    d_eligible_ = nullptr;
    d_succ_ = nullptr;
    d_procTimes_ = nullptr;
}

void JobShopGPUEvaluator::PrepareProblemDataGPU(const std::vector<JobShopData>& batch) {
    FreeProblemDataGPU(); // Frees d_problems_, d_jobs_, d_ops_ (reference problem data) etc.
    cpu_batch_data_ = JobShopDataGPU::PrepareBatchCPU(batch);
    num_problems_to_evaluate_ = batch.size();

    int num_problems_on_gpu_check = 0; // Renamed to avoid conflict
    JobShopDataGPU::UploadBatchToGPU(
        cpu_batch_data_, d_problems_, d_jobs_, d_ops_,
        d_eligible_, d_succ_, d_procTimes_, num_problems_on_gpu_check
    );
    if (num_problems_on_gpu_check != num_problems_to_evaluate_)
        throw std::runtime_error("Mismatch in number of problems uploaded to GPU for reference data.");

    // Now handle d_ops_working_
    if (num_problems_to_evaluate_ > 0) { // Only if there are problems to evaluate
        size_t required_total_elements = (size_t)nn_candidate_count_ * num_problems_to_evaluate_ * max_ops_per_problem_;
        size_t required_size_bytes = required_total_elements * sizeof(GPUOperation);

        if (required_size_bytes != current_d_ops_working_size_) {
            if (d_ops_working_ != nullptr) {
                CUDA_CHECK(hipFree(d_ops_working_));
            }
            CUDA_CHECK(hipMalloc(&d_ops_working_, required_size_bytes));
            current_d_ops_working_size_ = required_size_bytes;
        }

        // Populate d_ops_working_ by replicating problem data for each NN candidate
        std::vector<GPUOperation> h_ops_working_staging(required_total_elements);
        
        for (int nn_idx = 0; nn_idx < nn_candidate_count_; ++nn_idx) {
            for (int prob_idx_in_batch = 0; prob_idx_in_batch < num_problems_to_evaluate_; ++prob_idx_in_batch) {
                size_t dest_base_elem_offset = (nn_idx * num_problems_to_evaluate_ + prob_idx_in_batch) * max_ops_per_problem_;
                
                int src_ops_offset_in_batch_buffer = cpu_batch_data_.operationsOffsets[prob_idx_in_batch];
                int src_ops_count = cpu_batch_data_.operationsOffsets[prob_idx_in_batch + 1] - src_ops_offset_in_batch_buffer;

                if (src_ops_count > max_ops_per_problem_) {
                    // This is an issue: problem has more ops than allocated space per problem.
                    // Consider throwing an error or logging. For now, truncate (dangerous).
                    fprintf(stderr, "Warning: Problem %d has %d ops, exceeding max_ops_per_problem_ %d. Truncating.\n",
                            prob_idx_in_batch, src_ops_count, max_ops_per_problem_);
                    src_ops_count = max_ops_per_problem_;
                }
                
                if (src_ops_count > 0) {
                     memcpy(&h_ops_working_staging[dest_base_elem_offset], 
                            &cpu_batch_data_.operations[src_ops_offset_in_batch_buffer], 
                            src_ops_count * sizeof(GPUOperation));
                }

                // Zero out the remaining part of the slot for this problem if necessary
                if (src_ops_count < max_ops_per_problem_) {
                    memset(&h_ops_working_staging[dest_base_elem_offset + src_ops_count], 
                           0, 
                           (max_ops_per_problem_ - src_ops_count) * sizeof(GPUOperation));
                }
            }
        }
        CUDA_CHECK(hipMemcpy(d_ops_working_, h_ops_working_staging.data(), current_d_ops_working_size_, hipMemcpyHostToDevice));
    } else { // No problems to evaluate, free d_ops_working_ if it exists
        if (d_ops_working_ != nullptr) {
            CUDA_CHECK(hipFree(d_ops_working_));
            d_ops_working_ = nullptr;
            current_d_ops_working_size_ = 0;
        }
    }
}


bool JobShopGPUEvaluator::SetCurrentBatch(int batch_start, int batch_size) {
    auto t0 = std::chrono::high_resolution_clock::now();
    if (batch_start >= (int)cpu_problems_.size())
        return false;
    int batch_end = std::min(batch_start + batch_size, (int)cpu_problems_.size());
    std::vector<JobShopData> batch(cpu_problems_.begin() + batch_start, cpu_problems_.begin() + batch_end);
    auto t1 = std::chrono::high_resolution_clock::now();
    PrepareProblemDataGPU(batch);
    auto t2 = std::chrono::high_resolution_clock::now();
    std::cout << "[TIMER][CPU] Batch slicing: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count() << " ms, "
              << "PrepareProblemDataGPU: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() << " ms"
              << std::endl;
    return true;
}

__global__ void UpdateEvaluatorPointersKernel(
    NeuralNetwork::DeviceEvaluator* d_evaluators,
    float* d_all_weights,
    float* d_all_biases,
    int nn_total_weights_per_network,
    int nn_total_biases_per_network, // Add this parameter
    int nn_candidate_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nn_candidate_count) {
        d_evaluators[idx].weights = d_all_weights + (size_t)idx * nn_total_weights_per_network;
        d_evaluators[idx].biases  = d_all_biases  + (size_t)idx * nn_total_biases_per_network; // Corrected
    }
}


Eigen::VectorXd JobShopGPUEvaluator::EvaluateCandidates(const Eigen::MatrixXd& candidates, const bool& validation_mode) {
    auto t0 = std::chrono::high_resolution_clock::now();

    int nn_candidate_count = candidates.cols();
    if (candidates.rows() != nn_total_params_)
        throw std::runtime_error("Mismatch in number of weights per NN candidate.");

    auto t1 = std::chrono::high_resolution_clock::now();

    // 1. Populate Pinned Host Memory: Directly copy from Eigen matrix to the pinned host buffers
    for (int r = 0; r < nn_candidate_count; ++r) {
        int paramIdx = 0;
        size_t weight_offset = (size_t)r * nn_total_weights_per_network_;
        size_t bias_offset = (size_t)r * nn_total_biases_per_network_;
        for (size_t i = 1; i < nn_topology_.size(); ++i) {
            int prevLayerSize = nn_topology_[i - 1];
            int currLayerSize = nn_topology_[i];
            // Weights
            for (int w = 0; w < prevLayerSize * currLayerSize; ++w) {
                h_pinned_all_weights_[weight_offset++] = static_cast<float>(candidates(paramIdx++, r));
            }
            // Biases
            for (int b = 0; b < currLayerSize; ++b) {
                h_pinned_all_biases_[bias_offset++] = static_cast<float>(candidates(paramIdx++, r));
            }
        }
    }

    auto t2 = std::chrono::high_resolution_clock::now();

    // 2. Asynchronous Memory Transfer: Copy all weights and biases in single calls
    hipStream_t stream;
    hipStreamCreate(&stream);
    CUDA_CHECK(hipMemcpyAsync(d_all_candidate_weights_, h_pinned_all_weights_, total_weights_size_, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_all_candidate_biases_, h_pinned_all_biases_, total_biases_size_, hipMemcpyHostToDevice, stream));

    // Launch the pointer update kernel
    int threadsPerBlock = 128;
    int blocks = (nn_candidate_count + threadsPerBlock - 1) / threadsPerBlock;
    UpdateEvaluatorPointersKernel<<<blocks, threadsPerBlock, 0, stream>>>(
        d_evaluators_,
        d_all_candidate_weights_,
        d_all_candidate_biases_,
        nn_total_weights_per_network_,
        nn_total_biases_per_network_,
        nn_candidate_count
    );

    auto t3 = std::chrono::high_resolution_clock::now();

    float* d_results = nullptr;
    int result_count = nn_candidate_count;
    if (validation_mode) result_count = 1; // tylko jeden kandydat

    CUDA_CHECK(hipMalloc(&d_results, sizeof(float) * result_count));

    // Kernel
    auto t5 = std::chrono::high_resolution_clock::now();

    int kernel_blocks, kernel_threads;
    if (validation_mode) {
        kernel_threads = 256; // lub inna wielokrotność 32, np. 512
        kernel_blocks = (num_problems_to_evaluate_ + kernel_threads - 1) / kernel_threads;
    } else {
        kernel_blocks = nn_candidate_count_;
        kernel_threads = 192;
    }

    JobShopHeuristic::SolveBatchNew(
        d_problems_, d_evaluators_, d_ops_working_, d_results,
        num_problems_to_evaluate_,
        kernel_blocks,
        nn_total_weights_per_network_,
        nn_total_biases_per_network_,
        max_ops_per_problem_,
        stream,
        nn_total_params_,
        validation_mode
    );

    CUDA_CHECK(hipStreamSynchronize(stream));

    auto t6 = std::chrono::high_resolution_clock::now();

    std::vector<float> host_results(result_count);
    CUDA_CHECK(hipMemcpy(host_results.data(), d_results, sizeof(float) * result_count, hipMemcpyDeviceToHost));

    auto t7 = std::chrono::high_resolution_clock::now();

    Eigen::VectorXd fvalues(result_count);
    for (int r = 0; r < result_count; ++r)
        fvalues[r] = static_cast<double>(host_results[r]);

    auto t8 = std::chrono::high_resolution_clock::now();

    hipFree(d_results);
    hipStreamDestroy(stream);

    std::cout << "[TIMER][CPU] Weight Update : "
        << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() << " ms, "
        << "DeviceEvaluator H2D: "
        << std::chrono::duration_cast<std::chrono::milliseconds>(t3 - t2).count() << " ms, "
        << "Kernel launch+wait: "
        << std::chrono::duration_cast<std::chrono::milliseconds>(t6 - t5).count() << " ms, "
        << "Results D2H: "
        << std::chrono::duration_cast<std::chrono::milliseconds>(t7 - t6).count() << " ms, "
        << "fvalues fill: "
        << std::chrono::duration_cast<std::chrono::milliseconds>(t8 - t7).count() << " ms, "
        << "Total evaluateCandidates: "
        << std::chrono::duration_cast<std::chrono::milliseconds>(t8 - t0).count() << " ms"
        << std::endl;

    return fvalues;
}

float JobShopGPUEvaluator::EvaluateForMinMakespan(const Eigen::VectorXd& candidate_weights, int num_problems) {
    const int val_batch_size = 1000;
    int num_batches = (num_problems + val_batch_size - 1) / val_batch_size;
    float makespan_sum = 0.0f;
    int makespan_count = 0;

    for (int batch = 0; batch < num_batches; ++batch) {
        int batch_start = batch * val_batch_size;
        int batch_size = std::min(val_batch_size, num_problems - batch_start);

        if (!SetCurrentBatch(batch_start, batch_size)) {
            std::cerr << "[ERROR] Could not set batch for validation." << std::endl;
            continue;
        }

        Eigen::MatrixXd replicated_candidate_matrix(nn_total_params_, 1); // 1 candidate
        replicated_candidate_matrix.col(0) = candidate_weights;
        Eigen::VectorXd result_vector = EvaluateCandidates(replicated_candidate_matrix, true);

        makespan_sum += result_vector[0];
        makespan_count++;
    }

    if (makespan_count > 0)
        return makespan_sum / makespan_count;
    return std::numeric_limits<float>::max();
}
