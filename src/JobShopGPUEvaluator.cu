#include "hip/hip_runtime.h"
#include "JobShopGPUEvaluator.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstring>
#include <chrono>

JobShopGPUEvaluator::JobShopGPUEvaluator(const std::string& problem_file, const std::vector<int>& nn_topology)
    : nn_topology_(nn_topology)
{
    // all problems at once 
    cpu_problems_ = JobShopData::LoadFromParallelJson(problem_file, 100);//TODO fix nummber of problem assignment 
    if (cpu_problems_.empty())
        throw std::runtime_error("No problems loaded!");

    
    max_ops_per_problem_ = 0;
    for (const auto& prob : cpu_problems_) {
        int ops = 0;
        for (const auto& job : prob.jobs)
            ops += job.operations.size();
        if (ops > max_ops_per_problem_) max_ops_per_problem_ = ops;
    }

    nn_total_params_ = NeuralNetwork::CalculateTotalParameters(nn_topology_);
}

JobShopGPUEvaluator::~JobShopGPUEvaluator() {
    FreeProblemDataGPU();
}

void JobShopGPUEvaluator::FreeProblemDataGPU() {
    JobShopDataGPU::FreeBatchGPUData(d_problems_, d_jobs_, d_ops_, d_eligible_, d_succ_, d_procTimes_);
    d_problems_ = nullptr;
    d_jobs_ = nullptr;
    d_ops_ = nullptr;
    d_eligible_ = nullptr;
    d_succ_ = nullptr;
    d_procTimes_ = nullptr;
}

void JobShopGPUEvaluator::PrepareProblemDataGPU(const std::vector<JobShopData>& batch) {
    FreeProblemDataGPU();
    cpu_batch_data_ = JobShopDataGPU::PrepareBatchCPU(batch);
    num_problems_to_evaluate_ = batch.size();

    int num_problems_on_gpu = 0;
    JobShopDataGPU::UploadBatchToGPU(
        cpu_batch_data_, d_problems_, d_jobs_, d_ops_, 
        d_eligible_, d_succ_, d_procTimes_, num_problems_on_gpu
    );
    if (num_problems_on_gpu != num_problems_to_evaluate_)
        throw std::runtime_error("Mismatch in number of problems uploaded to GPU.");
}

bool JobShopGPUEvaluator::SetCurrentBatch(int batch_start, int batch_size) {
    auto t0 = std::chrono::high_resolution_clock::now();
    if (batch_start >= (int)cpu_problems_.size())
        return false;
    int batch_end = std::min(batch_start + batch_size, (int)cpu_problems_.size());
    std::vector<JobShopData> batch(cpu_problems_.begin() + batch_start, cpu_problems_.begin() + batch_end);
    auto t1 = std::chrono::high_resolution_clock::now();
    PrepareProblemDataGPU(batch);
    auto t2 = std::chrono::high_resolution_clock::now();
    std::cout << "[TIMER][CPU] Batch slicing: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count() << " ms, "
              << "PrepareProblemDataGPU: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() << " ms"
              << std::endl;
    return true;
}

Eigen::VectorXd JobShopGPUEvaluator::EvaluateCandidates(const Eigen::MatrixXd& candidates) {
    auto t0_total = std::chrono::high_resolution_clock::now();
    int nn_candidate_count = candidates.cols();
    if (candidates.rows() != nn_total_params_)
        throw std::runtime_error("Mismatch in number of weights per NN candidate.");

    auto t1_prep = std::chrono::high_resolution_clock::now();

    size_t single_nn_weights_count = 0;
    size_t single_nn_biases_count = 0;
    for (size_t i = 1; i < nn_topology_.size(); ++i) {
        single_nn_weights_count += nn_topology_[i-1] * nn_topology_[i];
        single_nn_biases_count += nn_topology_[i];
    }

    size_t total_weights_all_candidates_size = single_nn_weights_count * nn_candidate_count;
    size_t total_biases_all_candidates_size = single_nn_biases_count * nn_candidate_count;

    float* d_all_candidate_weights_mega_buffer = nullptr;
    float* d_all_candidate_biases_mega_buffer = nullptr;
    hipMalloc(&d_all_candidate_weights_mega_buffer, total_weights_all_candidates_size * sizeof(float));
    hipMalloc(&d_all_candidate_biases_mega_buffer, total_biases_all_candidates_size * sizeof(float));

    // int* d_topology_gpu = nullptr; // Remove this - d_topology is embedded in DeviceEvaluator
    // hipMalloc(&d_topology_gpu, nn_topology_.size() * sizeof(int)); // Remove this
    // hipMemcpy(d_topology_gpu, nn_topology_.data(), nn_topology_.size() * sizeof(int), hipMemcpyHostToDevice); // Remove this

    std::vector<NeuralNetwork::DeviceEvaluator> host_evaluators(nn_candidate_count);
    
    std::vector<float> temp_host_weights_buffer(single_nn_weights_count);
    std::vector<float> temp_host_biases_buffer(single_nn_biases_count);

    size_t current_mega_weight_offset_elements = 0;
    size_t current_mega_bias_offset_elements = 0;
    
    if (nn_topology_.size() > MAX_NN_LAYERS) { // MAX_NN_LAYERS should be accessible here or use a constant
        throw std::runtime_error("NN topology size exceeds MAX_NN_LAYERS");
    }

    for (int r = 0; r < nn_candidate_count; ++r) {
        int paramIdx = 0;
        size_t current_temp_weight_idx = 0;
        size_t current_temp_bias_idx = 0;
        
        for (size_t i = 1; i < nn_topology_.size(); ++i) {
            int prevLayerSize = nn_topology_[i-1];
            int currLayerSize = nn_topology_[i];
            
            for (int w = 0; w < prevLayerSize * currLayerSize; ++w)
                temp_host_weights_buffer[current_temp_weight_idx++] = static_cast<float>(candidates(paramIdx++, r));
            
            for (int b = 0; b < currLayerSize; ++b)
                temp_host_biases_buffer[current_temp_bias_idx++] = static_cast<float>(candidates(paramIdx++, r));
        }

        hipMemcpy(d_all_candidate_weights_mega_buffer + current_mega_weight_offset_elements,
                   temp_host_weights_buffer.data(),
                   single_nn_weights_count * sizeof(float),
                   hipMemcpyHostToDevice);
        
        hipMemcpy(d_all_candidate_biases_mega_buffer + current_mega_bias_offset_elements,
                   temp_host_biases_buffer.data(),
                   single_nn_biases_count * sizeof(float),
                   hipMemcpyHostToDevice);

        host_evaluators[r].d_weights = d_all_candidate_weights_mega_buffer + current_mega_weight_offset_elements;
        host_evaluators[r].d_biases = d_all_candidate_biases_mega_buffer + current_mega_bias_offset_elements;
        
        // Copy topology to embedded array
        memcpy(host_evaluators[r].d_topology, nn_topology_.data(), nn_topology_.size() * sizeof(int));
        for (size_t k_topo = nn_topology_.size(); k_topo < MAX_NN_LAYERS; ++k_topo) { // MAX_NN_LAYERS needs to be visible
            host_evaluators[r].d_topology[k_topo] = 0; // Pad with zeros
        }
        host_evaluators[r].num_layers = nn_topology_.size();

        current_mega_weight_offset_elements += single_nn_weights_count;
        current_mega_bias_offset_elements += single_nn_biases_count;
    }
    auto t2_eval_prep = std::chrono::high_resolution_clock::now();

    NeuralNetwork::DeviceEvaluator* d_evaluators = nullptr;
    hipMalloc(&d_evaluators, sizeof(NeuralNetwork::DeviceEvaluator) * nn_candidate_count);
    hipMemcpy(d_evaluators, host_evaluators.data(), sizeof(NeuralNetwork::DeviceEvaluator) * nn_candidate_count, hipMemcpyHostToDevice);
    auto t3_eval_h2d = std::chrono::high_resolution_clock::now();

    // Prepare ops working - This will be changed in Krok 3
    // For now, keep existing logic for ops_working, then replace
    std::vector<GPUOperation> ops_working(nn_candidate_count * num_problems_to_evaluate_ * max_ops_per_problem_);
    for (int w = 0; w < nn_candidate_count; ++w) {
      for (int p = 0; p < num_problems_to_evaluate_; ++p) {
        int base_idx = (w * num_problems_to_evaluate_ + p) * max_ops_per_problem_;
        int opsOffset = cpu_batch_data_.operationsOffsets[p]; 
        int opsCount = cpu_batch_data_.operationsOffsets[p+1] - cpu_batch_data_.operationsOffsets[p];
        
        if (opsCount > max_ops_per_problem_) {
            opsCount = max_ops_per_problem_; 
        }
        memcpy(&ops_working[base_idx], &cpu_batch_data_.operations[opsOffset], opsCount * sizeof(GPUOperation));
      }
    }
    auto t4_ops_memcpy_cpu = std::chrono::high_resolution_clock::now();

    GPUOperation* d_ops_working = nullptr;
    hipMalloc(&d_ops_working, ops_working.size() * sizeof(GPUOperation)); 
    hipMemcpy(d_ops_working, ops_working.data(), ops_working.size() * sizeof(GPUOperation), hipMemcpyHostToDevice);
    auto t5_ops_h2d = std::chrono::high_resolution_clock::now();
    
    float* d_results = nullptr;
    hipMalloc(&d_results, sizeof(float) * nn_candidate_count);

    auto t6_kernel_launch = std::chrono::high_resolution_clock::now();
    hipStream_t stream; 
    hipStreamCreate(&stream);

     JobShopHeuristic::SolveBatchNew(
        d_problems_, 
        d_evaluators, 
        // d_ops_, // This was the template, kernel accesses it via d_problems_
        d_ops_working, // Pass the allocated working memory for the kernel to use
        d_results, 
        num_problems_to_evaluate_, 
        nn_candidate_count, 
        max_ops_per_problem_, 
        // cpu_batch_data_.operationsOffsets.data(), // Not needed by this version of SolveBatchNew
        stream
    );

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream); 
    auto t7_kernel_sync = std::chrono::high_resolution_clock::now();

    std::vector<float> host_results(nn_candidate_count);
    hipMemcpy(host_results.data(), d_results, sizeof(float) * nn_candidate_count, hipMemcpyDeviceToHost);
    auto t8_results_d2h = std::chrono::high_resolution_clock::now();

    Eigen::VectorXd fvalues(nn_candidate_count);
    for (int r = 0; r < nn_candidate_count; ++r)
        fvalues[r] = static_cast<double>(host_results[r]);
    
    auto t9_fvalues_fill = std::chrono::high_resolution_clock::now();
    double min_makespan = (fvalues.size() > 0) ? fvalues.minCoeff() : 0.0;
    std::cout << "[INFO] Best average makespan: " << min_makespan << std::endl;

    hipFree(d_all_candidate_weights_mega_buffer);
    hipFree(d_all_candidate_biases_mega_buffer);
    // hipFree(d_topology_gpu); // Removed as d_topology is embedded
    hipFree(d_evaluators);
    hipFree(d_ops_working); 
    hipFree(d_results);
   
    // Update timers
    std::cout << "[TIMER][CPU] Evaluator NN Data Prep: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t2_eval_prep - t1_prep).count() << " ms, "
              << "Evaluator Struct H2D: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t3_eval_h2d - t2_eval_prep).count() << " ms, "
              << "ops_working CPU memcpy: " // This will be removed in Krok 3
              << std::chrono::duration_cast<std::chrono::milliseconds>(t4_ops_memcpy_cpu - t3_eval_h2d).count() << " ms, "
              << "ops_working H2D: " // This will be removed in Krok 3
              << std::chrono::duration_cast<std::chrono::milliseconds>(t5_ops_h2d - t4_ops_memcpy_cpu).count() << " ms, "
              << "Kernel launch+sync: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t7_kernel_sync - t6_kernel_launch).count() << " ms, "
              << "Results D2H: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t8_results_d2h - t7_kernel_sync).count() << " ms, "
              << "fvalues fill: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t9_fvalues_fill - t8_results_d2h).count() << " ms, "
              << "Total evaluateCandidates: "
              << std::chrono::duration_cast<std::chrono::milliseconds>(t9_fvalues_fill - t0_total).count() << " ms"
              << std::endl;

// Check the sizes
    int total_weights = 0;
    int total_biases = 0;
    for(int i = 1; i < nn_topology_.size(); i++) {
        total_weights += nn_topology_[i-1] * nn_topology_[i];
        total_biases += nn_topology_[i];
    }

    std::cout << "[DEBUG] Total weights calculated: " << total_weights << std::endl;
    std::cout << "[DEBUG] Total biases calculated: " << total_biases << std::endl;
    std::cout << "[DEBUG] Total parameters: " << nn_total_params_ << std::endl;

    return fvalues;
}