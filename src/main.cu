#include <cstdlib>
#include <ctime>
#include <iostream>
#include <vector>

#include "JobShopData.cuh"
#include "JobShopHeuristic.cuh"

int main() {
	srand(time(0));

	const bool generateRandomJobs = true;
	const bool generateRandomNNSetup = false;
	const int numProblems = 1;

	const std::vector<int> topology = {4, 32, 16, 1};

	try {
		// 1. Load or generate problem data
		JobShopData data;
		if(generateRandomJobs) {
			data = GenerateData();
			data.SaveToJson("jobshop_data");
		} else {
			data.LoadFromJson("jobshop_data");
		}

		// 2. Load or generate neural network
		NeuralNetwork nn;
		if(generateRandomNNSetup) {
			nn = NeuralNetwork(topology);
			nn.SaveToJson("weights_and_biases");
		} else {
			nn.LoadFromJson("weights_and_biases");
		}

		// 3. Prepare GPU data and upload to GPU

		auto solutions_batch = SolutionManager::CreateGPUSolutions(numProblems, data.numMachines, 100);

		GPUProblem* d_problems;
		std::vector<GPUProblem> h_problems(numProblems);

		GPUProblem template_problem = JobShopDataGPU::UploadToGPU(data);  // todo cleanup

		// Copy template to all problems
		hipMalloc(&d_problems, sizeof(GPUProblem) * numProblems);
		std::vector<GPUProblem> temp(numProblems, template_problem);
		hipMemcpy(d_problems, temp.data(),
				   sizeof(GPUProblem) * numProblems,
				   hipMemcpyHostToDevice);

		// 4. Create heuristic solver
		JobShopHeuristic heuristic(std::move(nn));
		
		// 5. Solve on GPU (even though we're just doing one problem)
		heuristic.SolveBatch(d_problems, &solutions_batch, numProblems);

		hipError_t kernelErr = hipGetLastError();
		if(kernelErr != hipSuccess) {
			std::cerr << "Kernel error: " << hipGetErrorString(kernelErr) << "\n";
		}

		// 6. Download results
		JobShopHeuristic::CPUSolution* solutions = new JobShopHeuristic::CPUSolution[numProblems];

		for(int i = 0; i < numProblems; ++i) {
			solutions[i].FromGPU(solutions_batch, i);
		}

		heuristic.PrintSchedule(solutions[0], data);

		// 7. Clean up GPU memory
		SolutionManager::FreeGPUSolutions(solutions_batch);
		hipFree(d_problems);
		JobShopDataGPU::FreeGPUData(template_problem);
		delete[] solutions;

	} catch(const std::exception& e) {
		std::cerr << "Error: " << e.what() << std::endl;
		return 1;
	}

	return 0;
}