#include <cstdlib>
#include <ctime>
#include <iostream>
#include <vector>

#include "JobShopData.cuh"
#include "JobShopHeuristic.cuh"

int main() {
    srand(time(0));

    const int numProblems = 100;
    std::vector<JobShopData> all_problems;
    const std::vector<int> topology = {4, 32, 16, 1};

    try {
        nlohmann::json j_array;
        {
            std::ifstream in(FileManager::GetFullPath("test_100.json"));
            if(!in) throw std::runtime_error("Failed to open file: test_100.json");
            in >> j_array;
            if(!j_array.is_array()) throw std::runtime_error("JSON root is not an array!");
            if(j_array.size() < numProblems) throw std::runtime_error("Not enough problems in file!");
        }
    
        
        std::vector<JobShopData> all_problems = JobShopData::LoadFromParallelJson("test_100.json", numProblems);
        // 2. Load neural network
        NeuralNetwork nn;
        nn.LoadFromJson("weights_and_biases");

        // 3. Prepare batch and upload to GPU
        auto batch = JobShopDataGPU::PrepareBatchCPU(all_problems);

        GPUProblem* d_problems = nullptr;
        GPUJob* d_jobs = nullptr;
        GPUOperation* d_ops = nullptr;
        int* d_eligible = nullptr;
        int* d_succ = nullptr;
        int* d_procTimes = nullptr;
        int numProblemsGPU = 0;

        JobShopDataGPU::UploadBatchToGPU(
            batch, d_problems, d_jobs, d_ops, d_eligible, d_succ, d_procTimes, numProblemsGPU);

        auto solutions_batch = SolutionManager::CreateGPUSolutions(numProblems, all_problems[0].numMachines, 100);

        // 4. Create heuristic solver
        JobShopHeuristic heuristic(std::move(nn));

        // 5. Solve on GPU
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        heuristic.SolveBatch(d_problems, &solutions_batch, numProblems);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

        hipEventDestroy(start);
        hipEventDestroy(stop);

        hipError_t kernelErr = hipGetLastError();
        if(kernelErr != hipSuccess) {
            std::cerr << "Kernel error: " << hipGetErrorString(kernelErr) << "\n";
        }

        // 6. Download results
        std::vector<JobShopHeuristic::CPUSolution> solutions(numProblems);
        for(int i = 0; i < numProblems; ++i) {
            solutions[i].FromGPU(solutions_batch, i);
        }

        heuristic.PrintSchedule(solutions[0], all_problems[0]);

        // 7. Clean up GPU memory
        SolutionManager::FreeGPUSolutions(solutions_batch);
        JobShopDataGPU::FreeBatchGPUData(d_problems, d_jobs, d_ops, d_eligible, d_succ, d_procTimes);

    } catch(const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    return 0;
}