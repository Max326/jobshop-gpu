#include <cstdlib>
#include <ctime>
#include <iostream>
#include <vector>

#include "JobShopData.cuh"
#include "JobShopHeuristic.cuh"

// TODO: implement more features, according to the Flexible Job Shop document
// 		- wasted time
// 		- total number of operations left
// 		- one hot encodings
// TODO: test scheduling correctness
// TODO: parallel operations (graphs)

int main() {
	srand(time(0));

	const bool useParallelData = true;
	const bool generateRandomJobs = false;
	const bool generateRandomNNSetup = false;
	const int numProblems = 10;

	std::vector<JobShopData> all_problems;



	const std::vector<int> topology = {4, 32, 16, 1};

	try {
		// 1. Load or generate problem data
		for (int i = 0; i < numProblems; ++i) {
			JobShopData data;
			data.LoadFromParallelJson("data_test.json", i);
			all_problems.push_back(std::move(data));
		}
		
		// 2. Load or generate neural network
		NeuralNetwork nn;
		if(generateRandomNNSetup) {
			nn = NeuralNetwork(topology);
			nn.SaveToJson("weights_and_biases");
		} else {
			nn.LoadFromJson("weights_and_biases");
		}

		// 3. Prepare GPU data and upload to GPU

		auto solutions_batch = SolutionManager::CreateGPUSolutions(numProblems, all_problems[0].numMachines, 100);

		GPUProblem* d_problems;
		std::vector<GPUProblem> h_problems(numProblems);

	

		for(int i=0; i<numProblems; ++i) {
			h_problems[i] = JobShopDataGPU::UploadToGPU(all_problems[i]);
		}

		// Copy template to all problems
		hipMalloc(&d_problems, sizeof(GPUProblem) * numProblems);

		// std::vector<GPUProblem> temp(numProblems, template_problem);

		hipMemcpy(d_problems, h_problems.data(),
				   sizeof(GPUProblem) * numProblems,
				   hipMemcpyHostToDevice);

		// 4. Create heuristic solver
		JobShopHeuristic heuristic(std::move(nn));
		
		// 5. Solve on GPU (even though we're just doing one problem)
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);
		heuristic.SolveBatch(d_problems, &solutions_batch, numProblems);
		hipEventRecord(stop);

		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

		hipEventDestroy(start);
		hipEventDestroy(stop);

		hipError_t kernelErr = hipGetLastError();
		if(kernelErr != hipSuccess) {
			std::cerr << "Kernel error: " << hipGetErrorString(kernelErr) << "\n";
		}

		// 6. Download results
		JobShopHeuristic::CPUSolution* solutions = new JobShopHeuristic::CPUSolution[numProblems];

		for(int i = 0; i < numProblems; ++i) {
			solutions[i].FromGPU(solutions_batch, i);
		}

		heuristic.PrintSchedule(solutions[0], all_problems[0]);

		// 7. Clean up GPU memory
		SolutionManager::FreeGPUSolutions(solutions_batch);
		hipFree(d_problems);

		// JobShopDataGPU::FreeGPUData(template_problem);

		for (int i = 0; i<numProblems; ++i) {
			JobShopDataGPU::FreeGPUData(h_problems[i]);
		}

		delete[] solutions;

	} catch(const std::exception& e) {
		std::cerr << "Error: " << e.what() << std::endl;
		return 1;
	}

	return 0;
}