#include "hip/hip_runtime.h"
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <vector>

#include "JobShopData.cuh"
#include "JobShopHeuristic.cuh"

int main() {
	srand(time(0));

	const bool generateRandomJobs = false;
	const bool generateRandomNNSetup = false;
	const int numProblems = 1;	// Start with 1 problem for testing

	const std::vector<int> topology = {4, 32, 16, 1};

	try {
		// 1. Load or generate problem data
		JobShopData data;
		if(generateRandomJobs) {
			data = GenerateData();
			data.SaveToJson("jobshop_data");
		} else {
			data.LoadFromJson("jobshop_data");
		}

		// 2. Load or generate neural network
		NeuralNetwork nn;
		if(generateRandomNNSetup) {
			nn = NeuralNetwork(topology);
			nn.SaveToJson("weights_and_biases");
		} else {
			nn.LoadFromJson("weights_and_biases");
		}

		// 3. Prepare GPU data
		// a) Upload the problem to GPU
		GPUProblem gpuProblem = JobShopDataGPU::UploadToGPU(data);

		// b) Create GPU solution container
		SolutionManager::GPUSolution gpuSolution =
			SolutionManager::CreateGPUSolution(data.numMachines, 100);	// ! 100 ops per machine max -- this needs to be the same in JopShopHeuristic.cuh

		// 4. Create heuristic solver
		JobShopHeuristic heuristic(std::move(nn));

		// 5. Solve on GPU (even though we're just doing one problem)
		heuristic.SolveBatch(&gpuProblem, &gpuSolution, numProblems);

		hipError_t kernelErr = hipGetLastError();
		if(kernelErr != hipSuccess) {
			std::cerr << "Kernel error: " << hipGetErrorString(kernelErr) << "\n";
		}

		// 6. Download and display results
		JobShopHeuristic::CPUSolution solution;
		solution.FromGPU(gpuSolution);
		heuristic.PrintSchedule(solution, data);

		// 7. Clean up GPU memory
		SolutionManager::FreeGPUSolution(gpuSolution);
		JobShopDataGPU::FreeGPUData(gpuProblem);

	} catch(const std::exception& e) {
		std::cerr << "Error: " << e.what() << std::endl;
		return 1;
	}

	return 0;
}